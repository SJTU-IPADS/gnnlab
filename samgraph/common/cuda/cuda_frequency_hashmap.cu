#include "hip/hip_runtime.h"
/*
 * Copyright 2022 Institute of Parallel and Distributed Systems, Shanghai Jiao Tong University
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>

#include "../constant.h"
#include "../device.h"
#include "../logging.h"
#include "../profiler.h"
#include "../timer.h"
#include "cuda_frequency_hashmap.h"
#include "cuda_utils.h"

namespace samgraph {
namespace common {
namespace cuda {

namespace {

size_t TableSize(const size_t num, const size_t scale) {
  /** SXN: why bother? 1+[log2([n/2])] is similar to [log2(n)]*/
  const size_t next_pow2 = 1 << static_cast<size_t>(1 + std::log2(num >> 1));
  return next_pow2 << scale;
}

class MutableDeviceFrequencyHashmap : public DeviceFrequencyHashmap {
 public:
  typedef typename DeviceFrequencyHashmap::NodeBucket *NodeIterator;
  typedef typename DeviceFrequencyHashmap::EdgeBucket *EdgeIterator;

  explicit MutableDeviceFrequencyHashmap(FrequencyHashmap *const host_map)
      : DeviceFrequencyHashmap(host_map->DeviceHandle()) {}

  inline __device__ NodeIterator SearchNode(const IdType id) {
#ifndef SXN_REVISED
    const IdType pos = SearchNodeForPosition(id);
    return GetMutableNode(pos);
#else
    return GetMutableNode(id);
#endif
  }

  inline __device__ EdgeIterator SearchEdge(const IdType node_idx,
                                            const IdType dst) {
    const IdType pos = SearchEdgeForPosition(node_idx, dst);
    return GetMutableEdge(pos);
  }

#ifndef SXN_REVISED
  inline __device__ bool AttemptInsertNodeAt(const IdType pos,
                                             const IdType id) {
    const IdType key =
        atomicCAS(&GetMutableNode(pos)->key, Constant::kEmptyKey, id);
    if (key == Constant::kEmptyKey || key == id) {
      // printf("Insert %u insert at pos %u\n", id, pos);
      return true;
    } else {
      return false;
    }
  }

  inline __device__ NodeIterator InsertNode(const IdType id) {
    IdType pos = NodeHash(id);

    IdType delta = 1;
    while (!AttemptInsertNodeAt(pos, id)) {
      // printf("node %u insert pos %u fail\n", id, pos);
      pos = NodeHash(pos + delta);
      delta += 1;
    }
    return GetMutableNode(pos);
  }
#endif

  inline __device__ bool AttemptInsertEdgeAt(const IdType pos, const IdType src,
                                             const IdType dst,
                                             const IdType index) {
    EdgeIterator edge_iter = GetMutableEdge(pos);
    const IdType key = atomicCAS(&edge_iter->key, Constant::kEmptyKey, dst);
    if (key == Constant::kEmptyKey || key == dst) {
      atomicAdd(&edge_iter->count, 1U);
#ifndef SXN_REVISED
      atomicCAS(&edge_iter->index, Constant::kEmptyKey, index);
      if (key == Constant::kEmptyKey) {
        NodeIterator node_iter = SearchNode(src);
        atomicAdd(&node_iter->count, 1U);
      }
#else
      /** SXN: remove atomic by checking swapped out key */
      if (key == Constant::kEmptyKey) {
        edge_iter->index = index;
        NodeIterator node_iter = SearchNode(PosToNodeIdx(pos));
        atomicAdd(node_iter, 1U);
      }
#endif
      return true;
    } else {
      return false;
    }
  }

  inline __device__ EdgeIterator InsertEdge(const IdType node_idx,
                                            const IdType src, const IdType dst,
                                            const IdType index) {
    IdType start_off = node_idx * _per_node_etable_size;
    IdType pos = EdgeHash(dst);

    IdType delta = 1;
    while (!AttemptInsertEdgeAt(start_off + pos, src, dst, index)) {
      pos = EdgeHash(pos + delta);
      delta += 1;
    }

    return GetMutableEdge(start_off + pos);
  }

  inline __device__ NodeIterator GetMutableNode(const IdType pos) {
    assert(pos < _ntable_size);
    return const_cast<NodeIterator>(_node_table + pos);
  }

  inline __device__ EdgeIterator GetMutableEdge(const IdType pos) {
    assert(pos < _etable_size);
    return const_cast<EdgeIterator>(_edge_table + pos);
  }

  inline __device__ IdType GetRelativePos(const EdgeIterator iter) {
    return iter - _edge_table;
  }
};

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void init_node_table(MutableDeviceFrequencyHashmap table,
                                const size_t num_bucket) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using NodeIterator = typename MutableDeviceFrequencyHashmap::NodeIterator;

#pragma unroll
  for (IdType index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_bucket) {
      NodeIterator node_iter = table.GetMutableNode(index);
#ifndef SXN_REVISED
      node_iter->key = Constant::kEmptyKey;
      node_iter->count = 0;
#else
      *node_iter = 0;
#endif
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void init_edge_table(MutableDeviceFrequencyHashmap table,
                                const size_t num_bucket) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using EdgeIterator = typename MutableDeviceFrequencyHashmap::EdgeIterator;

#pragma unroll
  for (IdType index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_bucket) {
      EdgeIterator edge_iter = table.GetMutableEdge(index);
      edge_iter->key = Constant::kEmptyKey;
      edge_iter->count = 0;
      edge_iter->index = Constant::kEmptyKey;
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void init_unique_range(IdType *_unique_range,
                                  const size_t unique_list_size) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using EdgeIterator = typename MutableDeviceFrequencyHashmap::EdgeIterator;

#pragma unroll
  for (IdType index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < unique_list_size) {
      _unique_range[index] = index;
    }
  }
}

#ifndef SXN_REVISED
template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void reset_node_table(MutableDeviceFrequencyHashmap table,
                                 const IdType *nodes, const size_t num_nodes) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using NodeIterator = typename MutableDeviceFrequencyHashmap::NodeIterator;

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_nodes) {
      IdType id = nodes[index];
      NodeIterator node_iter = table.SearchNode(id);
      node_iter->key = Constant::kEmptyKey;
      node_iter->count = 0;
    }
  }
}
#else
template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void reset_node_table(MutableDeviceFrequencyHashmap table,
                                 const IdType *nodes, const size_t num_nodes) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using NodeIterator = typename MutableDeviceFrequencyHashmap::NodeIterator;

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_nodes) {
      NodeIterator node_iter = table.SearchNode(index);
      *node_iter = 0;
    }
  }
}
#endif

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void reset_edge_table(MutableDeviceFrequencyHashmap table,
                                 IdType *unique_node_idx, IdType *unique_dst,
                                 const size_t num_unique) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using EdgeIterator = typename MutableDeviceFrequencyHashmap::EdgeIterator;

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_unique) {
      IdType node_idx = unique_node_idx[index];
      IdType dst = unique_dst[index];
      EdgeIterator edge_iter = table.SearchEdge(node_idx, dst);
      edge_iter->key = Constant::kEmptyKey;
      edge_iter->count = 0;
      edge_iter->index = Constant::kEmptyKey;
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void reset_edge_table_revised(MutableDeviceFrequencyHashmap table,
                                 IdType *unique_pos, IdType *unique_dst,
                                 const size_t num_unique) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using EdgeIterator = typename MutableDeviceFrequencyHashmap::EdgeIterator;

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_unique) {
      IdType dst = unique_dst[index];
      EdgeIterator edge_iter = table.GetMutableEdge(unique_pos[index]);
      assert(edge_iter->key == dst);
      edge_iter->key = Constant::kEmptyKey;
      edge_iter->count = 0;
      edge_iter->index = Constant::kEmptyKey;
    }
  }
}

#ifndef SXN_REVISED
template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void populate_node_table(const IdType *nodes,
                                    const size_t num_input_node,
                                    MutableDeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_input_node) {
      table.InsertNode(nodes[index]);
    }
  }
}
#else
#endif

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void count_frequency(const IdType *input_src,
                                const IdType *input_dst,
                                const size_t num_input_edge,
                                const size_t edges_per_node,
                                IdType *item_prefix,
                                MutableDeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using BlockReduce = typename hipcub::BlockReduce<IdType, BLOCK_SIZE>;
  using EdgeIterator = typename MutableDeviceFrequencyHashmap::EdgeIterator;

  IdType count = 0;
#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_input_edge && input_src[index] != Constant::kEmptyKey) {
      IdType node_idx = index / edges_per_node;
      EdgeIterator edge_iter =
          table.InsertEdge(node_idx, input_src[index], input_dst[index], index);
      if (edge_iter->index == index) {
        ++count;
      }
    }
  }

  __shared__ typename BlockReduce::TempStorage temp_space;

  count = BlockReduce(temp_space).Sum(count);

  if (threadIdx.x == 0) {
    item_prefix[blockIdx.x] = count;
    if (blockIdx.x == 0) {
      item_prefix[gridDim.x] = 0;
    }
  }
}
template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void count_frequency_revised(
                                IdType *input_src,
                                IdType *input_dst,
                                const size_t num_input_edge,
                                const size_t edges_per_node,
                                IdType *item_prefix,
                                MutableDeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using BlockReduce = typename hipcub::BlockReduce<IdType, BLOCK_SIZE>;
  using EdgeIterator = typename MutableDeviceFrequencyHashmap::EdgeIterator;

  IdType count = 0;
#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_input_edge && input_src[index] != Constant::kEmptyKey) {
      IdType node_idx = index / edges_per_node;
      EdgeIterator edge_iter =
          table.InsertEdge(node_idx, input_src[index], input_dst[index], index);
      input_src[index] = Constant::kEmptyKey;
      if (edge_iter->index == index) {
        input_src[index] = table.GetRelativePos(edge_iter);
        ++count;
      }
    }
  }

  __shared__ typename BlockReduce::TempStorage temp_space;

  count = BlockReduce(temp_space).Sum(count);

  if (threadIdx.x == 0) {
    item_prefix[blockIdx.x] = count;
    if (blockIdx.x == 0) {
      item_prefix[gridDim.x] = 0;
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void generate_unique_edges(
    const IdType *input_src, const IdType *input_dst,
    const size_t num_input_edge, IdType *item_prefix, IdType *unique_node_idx,
    IdType *unique_src, IdType *unique_dst, IdType *unique_count,
    size_t *num_unique, const size_t edges_per_node,
    MutableDeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);

  using FlagType = IdType;
  using BlockScan = typename hipcub::BlockScan<FlagType, BLOCK_SIZE>;
  using EdgeBucket = typename DeviceFrequencyHashmap::EdgeBucket;

  constexpr const IdType VALS_PER_THREAD = TILE_SIZE / BLOCK_SIZE;

  __shared__ typename BlockScan::TempStorage temp_space;

  const IdType offset = item_prefix[blockIdx.x];

  BlockPrefixCallbackOp<FlagType> prefix_op(0);

  // count successful placements
  for (IdType i = 0; i < VALS_PER_THREAD; ++i) {
    const IdType index = threadIdx.x + i * BLOCK_SIZE + blockIdx.x * TILE_SIZE;

    IdType node_idx = index / edges_per_node;
    FlagType flag;
    EdgeBucket *bucket;
    if (index < num_input_edge && input_src[index] != Constant::kEmptyKey) {
      bucket = table.SearchEdge(node_idx, input_dst[index]);
      flag = bucket->index == index;
    } else {
      flag = 0;
    }

    if (!flag) {
      bucket = nullptr;
    }

    BlockScan(temp_space).ExclusiveSum(flag, flag, prefix_op);
    __syncthreads();

    if (bucket) {
      const IdType pos = offset + flag;
      unique_node_idx[pos] = node_idx;
      unique_src[pos] = input_src[index];
      unique_dst[pos] = input_dst[index];
      unique_count[pos] = bucket->count;
    }
  }

  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *num_unique = item_prefix[gridDim.x];
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void generate_unique_edges_pos(
    IdType *input_src,
    const IdType *input_nodes,
    const size_t num_input_node,
    const size_t num_input_edge, IdType *item_prefix,
    Id64Type * unique_combination_key,
    IdType *unique_edge_pos, const size_t edges_per_node,
    MutableDeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);

  using FlagType = IdType;
  using BlockScan = typename hipcub::BlockScan<FlagType, BLOCK_SIZE>;
  using EdgeBucket = typename DeviceFrequencyHashmap::EdgeBucket;

  constexpr const IdType VALS_PER_THREAD = TILE_SIZE / BLOCK_SIZE;

  __shared__ typename BlockScan::TempStorage temp_space;

  const IdType offset = item_prefix[blockIdx.x];

  BlockPrefixCallbackOp<FlagType> prefix_op(0);

  // count successful placements
  for (IdType i = 0; i < VALS_PER_THREAD; ++i) {
    const IdType index = threadIdx.x + i * BLOCK_SIZE + blockIdx.x * TILE_SIZE;

    IdType node_idx = index / edges_per_node;
    FlagType flag = 0;
    EdgeBucket *bucket = nullptr;
    if (index < num_input_edge && input_src[index] != Constant::kEmptyKey) {
      /** SXN: optimize: input dst can be modified to location in hash_table,
       * thus no need to search */
      bucket = table.GetMutableEdge(input_src[index]);
      flag = 1;
    }

    BlockScan(temp_space).ExclusiveSum(flag, flag, prefix_op);
    __syncthreads();

    if (bucket) {
      const IdType pos = offset + flag;
      unique_edge_pos[pos] = table.GetRelativePos(bucket);
      unique_combination_key[pos] = 
          (Id64Type)((num_input_node - node_idx) << 32) | 
          ((Id64Type)bucket->count); 
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void construct_unique_edge_list(
    const Id64Type *unique_combination_key,
    const IdType *unique_pos,
    IdType *unique_node_idx,
    IdType *unique_dst,
    const size_t num_unique,
    MutableDeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_unique) {
      unique_node_idx[index] = table.PosToNodeIdx(unique_pos[index]);
      unique_dst[index] = table.GetMutableEdge(unique_pos[index])->key;
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void reorder_unique(
    const IdType *unique_src, const IdType *unique_idx,
    const IdType *tmp_unique_node_idx, const IdType *tmp_unique_dst,
    const IdType *tmp_unique_frequency, IdType *unique_node_idx,
    IdType *unique_dst, IdType *unique_frequency,
    Id64Type *unique_combination_key, const size_t num_unique) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_unique) {
      IdType origin_idx = unique_idx[index];
      unique_node_idx[index] = tmp_unique_node_idx[origin_idx];
      unique_dst[index] = tmp_unique_dst[origin_idx];
      unique_frequency[index] = tmp_unique_frequency[origin_idx];
      unique_combination_key[index] =
          (((Id64Type)unique_src[index]) << 32) |
          ((Id64Type)tmp_unique_frequency[origin_idx]);
    }
  }
}
#ifndef SXN_REVISED
template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void generate_num_edge(const IdType *nodes, const size_t num_nodes,
                                  const size_t K, IdType *num_edge_prefix,
                                  IdType *num_output_prefix,
                                  DeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using NodeBucket = typename DeviceFrequencyHashmap::NodeBucket;

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_nodes) {
      const NodeBucket &bucket = *table.SearchNode(nodes[index]);
      num_edge_prefix[index] = bucket.count;
      num_output_prefix[index] = bucket.count > K ? K : bucket.count;
    }
  }

  if (threadIdx.x == 0 && blockIdx.x == 0) {
    num_edge_prefix[num_nodes] = 0;
    num_output_prefix[num_nodes] = 0;
  }
}
#else
template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void generate_num_edge(const IdType *, const size_t num_nodes,
                                  const size_t K, IdType *,
                                  IdType *num_output_prefix,
                                  DeviceFrequencyHashmap table) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using NodeBucket = typename DeviceFrequencyHashmap::NodeBucket;

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_nodes) {
      const NodeBucket &count = *table.SearchNode(index);
      num_output_prefix[index] = count > K ? K : count;
    }
  }

  if (threadIdx.x == 0 && blockIdx.x == 0) {
    num_output_prefix[num_nodes] = 0;
  }
}
#endif

__global__ void compact_output(const IdType *unique_src,
                               const IdType *unique_dst,
                               const IdType *unique_frequency,
                               const size_t num_nodes, const size_t K,
                               const IdType *num_unique_prefix,
                               const IdType *num_output_prefix,
                               IdType *output_src, IdType *output_dst,
                               IdType *output_data, size_t *num_output) {
  size_t i = blockIdx.x * blockDim.y + threadIdx.y;
  const size_t stride = blockDim.y * gridDim.x;

  /** SXN: this loop `may` be unnecessary */
  while (i < num_nodes) {
    IdType k = threadIdx.x;
    IdType max_output = num_output_prefix[i + 1] - num_output_prefix[i];
    /** SXN: max_output must <= K, ensured in generate_num_edge */
    while (k < K && k < max_output) {
      IdType from_off = num_unique_prefix[i] + k;
      IdType to_off = num_output_prefix[i] + k;

      output_src[to_off] = unique_src[from_off];
      output_dst[to_off] = unique_dst[from_off];
      output_data[to_off] = unique_frequency[from_off];

      k += blockDim.x;
    }

    i += stride;
  }

  if (blockIdx.x == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
    *num_output = num_output_prefix[num_nodes];
  }
}
__global__ void compact_output_revised(
    const IdType* input_nodes,
    const Id64Type *unique_combination_key,
    const IdType *unique_dst,
    const size_t num_nodes, const size_t K,
    const IdType *num_unique_prefix,
    const IdType *num_output_prefix,
    IdType *output_src, IdType *output_dst,
    IdType *output_data, size_t *num_output) {
  size_t i = blockIdx.x * blockDim.y + threadIdx.y;

  /** SXN: this loop `may` be unnecessary */
  if (i < num_nodes) {
    IdType k = threadIdx.x;
    IdType max_output = num_output_prefix[i + 1] - num_output_prefix[i];
    /** SXN: max_output must <= K, ensured in generate_num_edge */
    while (k < max_output) {
      IdType from_off = num_unique_prefix[i] + k;
      IdType to_off = num_output_prefix[i] + k;
      // IdType src_node_idx = num_nodes - (unique_combination_key[from_off] >> 32);
      // assert(src_node_idx == i);
      output_src[to_off] = input_nodes[i];
      output_data[to_off] = (unique_combination_key[from_off]);
      output_dst[to_off] = unique_dst[from_off];

      k += blockDim.x;
    }
  }

  if (blockIdx.x == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
    *num_output = num_output_prefix[num_nodes];
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void validate_combine_w_freq(const IdType *unique_src,
                                        IdType *unique_frequency,
                                        Id64Type *unique_combination_key,
                                        const size_t num_unique) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_unique) {
      IdType original_node_id = unique_src[index];
      Id64Type ck = unique_combination_key[index];
      IdType extracted_node_id = (ck >> 32) & 0x00000000ffffffff;
      IdType extracted_freq = ck & 0x00000000ffffffff;
      assert(extracted_node_id == original_node_id);
      assert(extracted_freq == unique_frequency[index]);
    }
  }
}
template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void validate_combine(const IdType *unique_src,
                                 Id64Type *unique_combination_key,
                                 const size_t num_unique) {
  assert(BLOCK_SIZE == blockDim.x);
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_unique) {
      IdType original_node_id = unique_src[index];
      Id64Type ck = unique_combination_key[index];
      IdType extracted_node_id = (ck >> 32) & 0x00000000ffffffff;
      assert(extracted_node_id == original_node_id);
    }
  }
}

}  // namespace

DeviceFrequencyHashmap::DeviceFrequencyHashmap(
    const NodeBucket *node_table, const EdgeBucket *edge_table,
    const size_t ntable_size, const size_t etable_size,
    const size_t per_node_etable_size)
    : _node_table(node_table),
      _edge_table(edge_table),
      _ntable_size(ntable_size),
      _etable_size(etable_size),
      _per_node_etable_size(per_node_etable_size) {}

DeviceFrequencyHashmap FrequencyHashmap::DeviceHandle() const {
  return DeviceFrequencyHashmap(_node_table, _edge_table, _ntable_size,
                                _etable_size, _per_node_etable_size);
}

FrequencyHashmap::FrequencyHashmap(const size_t max_nodes,
                                   const size_t edges_per_node, Context ctx,
                                   const size_t node_table_scale,
                                   const size_t edge_table_scale)
    : _ctx(ctx),
      _max_nodes(max_nodes),
      _edges_per_node(edges_per_node),
#ifndef SXN_REVISED
      _ntable_size(TableSize(max_nodes, node_table_scale)),
#else
      _ntable_size(max_nodes + 1),
#endif
      _etable_size(max_nodes * TableSize(edges_per_node, edge_table_scale)),
      _per_node_etable_size(TableSize(edges_per_node, edge_table_scale)),
      _num_node(0),
      _node_list_size(max_nodes),
      _num_unique(0),
      _unique_list_size(max_nodes * edges_per_node) {
  auto device = Device::Get(_ctx);
  CHECK_EQ(_ctx.device_type, kGPU);

  _node_table = static_cast<NodeBucket *>(
      device->AllocDataSpace(_ctx, sizeof(NodeBucket) * _ntable_size));
  _edge_table = static_cast<EdgeBucket *>(
      device->AllocDataSpace(_ctx, sizeof(EdgeBucket) * _etable_size));

#ifndef SXN_REVISED
  _node_list = static_cast<IdType *>(
      device->AllocDataSpace(_ctx, sizeof(IdType) * _node_list_size));

  _unique_range = static_cast<IdType *>(
      device->AllocDataSpace(_ctx, sizeof(IdType) * _unique_list_size));
  _unique_node_idx = static_cast<IdType *>(
      device->AllocDataSpace(_ctx, sizeof(IdType) * _unique_list_size));
  _unique_src = static_cast<IdType *>(
      device->AllocDataSpace(_ctx, sizeof(IdType) * _unique_list_size));
  _unique_dst = static_cast<IdType *>(
      device->AllocDataSpace(_ctx, sizeof(IdType) * _unique_list_size));
  _unique_frequency = static_cast<IdType *>(
      device->AllocDataSpace(_ctx, sizeof(IdType) * _unique_list_size));
  _unique_combination_key = static_cast<Id64Type *>(
      device->AllocDataSpace(_ctx, sizeof(Id64Type) * _unique_list_size));
#else
  _unique_node_idx = static_cast<IdType *>(
      device->AllocDataSpace(_ctx, sizeof(IdType) * _unique_list_size * 2));
  _unique_dst = _unique_node_idx + _unique_list_size;
  _unique_combination_key = static_cast<Id64Type *>(
      device->AllocDataSpace(_ctx, sizeof(Id64Type) * _unique_list_size));
#endif

  auto device_table = MutableDeviceFrequencyHashmap(this);
  dim3 grid0(RoundUpDiv(_ntable_size, Constant::kCudaTileSize));
  dim3 grid1(RoundUpDiv(_etable_size, Constant::kCudaTileSize));
  dim3 block0(Constant::kCudaBlockSize);
  dim3 block1(Constant::kCudaBlockSize);

  init_node_table<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid0, block0>>>(device_table, _ntable_size);
  init_edge_table<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid1, block1>>>(device_table, _etable_size);
#ifndef SXN_REVISED
  dim3 grid2(RoundUpDiv(_unique_list_size, Constant::kCudaTileSize));
  dim3 block2(Constant::kCudaBlockSize);
  init_unique_range<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid2, block2>>>(_unique_range, _unique_list_size);
#else
#endif

  LOG(INFO) << "FrequencyHashmap init with node table size: " << _ntable_size
            << " and edge table size: " << _etable_size;
}

FrequencyHashmap::~FrequencyHashmap() {
  auto device = Device::Get(_ctx);

  device->FreeDataSpace(_ctx, _node_table);
  device->FreeDataSpace(_ctx, _edge_table);
#ifndef SXN_REVISED
  device->FreeDataSpace(_ctx, _node_list);
  device->FreeDataSpace(_ctx, _unique_range);
  device->FreeDataSpace(_ctx, _unique_node_idx);
  device->FreeDataSpace(_ctx, _unique_src);
  device->FreeDataSpace(_ctx, _unique_dst);
  device->FreeDataSpace(_ctx, _unique_frequency);
#else
  device->FreeDataSpace(_ctx, _unique_node_idx);
#endif
  device->FreeDataSpace(_ctx, _unique_combination_key);
}

#ifndef SXN_REVISED
void FrequencyHashmap::GetTopK(const IdType *input_src, const IdType *input_dst,
                               const size_t num_input_edge,
                               const IdType *input_nodes,
                               const size_t num_input_node, const size_t K,
                               IdType *output_src, IdType *output_dst,
                               IdType *output_data, size_t *num_output,
                               StreamHandle stream, uint64_t task_key) {
  const size_t num_tiles0 = RoundUpDiv(num_input_node, Constant::kCudaTileSize);
  const size_t num_tiles1 = RoundUpDiv(num_input_edge, Constant::kCudaTileSize);
  const dim3 grid0(num_tiles0);
  const dim3 grid1(num_tiles1);

  const dim3 block0(Constant::kCudaBlockSize);
  const dim3 block1(Constant::kCudaBlockSize);

  dim3 block2(Constant::kCudaBlockSize, 1);
  while (static_cast<size_t>(block2.x) >= 2 * K) {
    block2.x /= 2;
    block2.y *= 2;
  }
  dim3 grid2(RoundUpDiv(num_input_node, static_cast<size_t>(block2.y)));

  auto device_table = MutableDeviceFrequencyHashmap(this);
  auto device = Device::Get(_ctx);
  auto cu_stream = static_cast<hipStream_t>(stream);

  size_t workspace_bytes0;
  size_t workspace_bytes1;
  size_t workspace_bytes2;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, workspace_bytes0, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), grid0.x + 1, cu_stream));
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, workspace_bytes1, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), grid1.x + 1, cu_stream));
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, workspace_bytes2, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), num_input_node + 1, cu_stream));
  device->StreamSync(_ctx, stream);

  void *workspace0 = device->AllocWorkspace(_ctx, workspace_bytes0);
  void *workspace1 = device->AllocWorkspace(_ctx, workspace_bytes1);
  void *workspace2 = device->AllocWorkspace(_ctx, workspace_bytes2);

  // 1. populate the node table
  Timer t1;
  populate_node_table<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid0, block0, 0, cu_stream>>>(input_nodes, num_input_node,
                                        device_table);
  device->StreamSync(_ctx, stream);
  double step1_time = t1.Passed();

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 1 finish with "
             << num_input_node << " input nodes with grid " << grid0.x
             << " block " << block0.x;

  // 2. count frequency of every unique edge and
  //    count unique edges for every node
  Timer t2;
  IdType *num_unique_prefix = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, sizeof(IdType) * (grid1.x + 1)));
  count_frequency<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid1, block1, 0, cu_stream>>>(input_src, input_dst, num_input_edge,
                                        _edges_per_node, num_unique_prefix,
                                        device_table);
  device->StreamSync(_ctx, stream);
  double step2_time = t2.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 2 finish";

  // 3. count the number of unique edges.
  //    prefix sum the the array
  Timer t3;

  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(workspace1, workspace_bytes1,
                                          num_unique_prefix, num_unique_prefix,
                                          grid1.x + 1, cu_stream));
  device->StreamSync(_ctx, stream);
  double step3_time = t3.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 3 finish";

  // 4. get the array of all unique edges.
  Timer t4;
  size_t *device_num_unique =
      static_cast<size_t *>(device->AllocWorkspace(_ctx, sizeof(size_t)));
  IdType *tmp_unique_node_idx = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, sizeof(IdType) * _unique_list_size));
  IdType *tmp_unique_dst = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, sizeof(IdType) * _unique_list_size));
  IdType *tmp_unique_frequency = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, sizeof(IdType) * _unique_list_size));
  generate_unique_edges<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid1, block1, 0, cu_stream>>>(input_src, input_dst, num_input_edge,
                                        num_unique_prefix, tmp_unique_node_idx,
                                        _unique_src, tmp_unique_dst,
                                        tmp_unique_frequency, device_num_unique,
                                        _edges_per_node, device_table);
  device->StreamSync(_ctx, stream);

  /** SXN: last item of num_unique_prefix. no need to use another
   * device_num_unique */
  device->CopyDataFromTo(device_num_unique, 0, &_num_unique, 0, sizeof(size_t),
                         _ctx, CPU(), stream);
  device->StreamSync(_ctx, stream);

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 4 finish with number of unique "
             << _num_unique;
  double step4_time = t4.Passed();

  // 5. pair-sort unique array using src as key.
  Timer t5;

  IdType *unique_idx = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, sizeof(IdType) * _num_unique));
  device->CopyDataFromTo(_unique_range, 0, unique_idx, 0,
                         sizeof(IdType) * _num_unique, _ctx, _ctx, stream);
  device->StreamSync(_ctx, stream);

  size_t workspace_bytes3;
  CUDA_CALL(hipcub::DeviceRadixSort::SortPairsDescending(
      nullptr, workspace_bytes3, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), _num_unique, 0, sizeof(IdType) * 8,
      cu_stream));
  device->StreamSync(_ctx, stream);

  void *workspace3 = device->AllocWorkspace(_ctx, workspace_bytes3);
  CUDA_CALL(hipcub::DeviceRadixSort::SortPairsDescending(
      workspace3, workspace_bytes3, _unique_src, _unique_src, unique_idx,
      unique_idx, _num_unique, 0, sizeof(IdType) * 8, cu_stream));
  device->StreamSync(_ctx, stream);

  const size_t num_tiles3 = RoundUpDiv(_num_unique, Constant::kCudaTileSize);
  const dim3 grid3(num_tiles3);
  const dim3 block3(Constant::kCudaBlockSize);

  reorder_unique<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid3, block3, 0, cu_stream>>>(
          _unique_src, unique_idx, tmp_unique_node_idx, tmp_unique_dst,
          tmp_unique_frequency, _unique_node_idx, _unique_dst,
          _unique_frequency, _unique_combination_key, _num_unique);
  device->StreamSync(_ctx, stream);
  double step5_time = t5.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 5 finish";

  // 6. sort the unique src node array.
  Timer t6;
  device->CopyDataFromTo(input_nodes, 0, _node_list, 0,
                         num_input_node * sizeof(IdType), _ctx, _ctx, stream);
  device->StreamSync(_ctx, stream);
  _num_node = num_input_node;

  size_t workspace_bytes4;
  CUDA_CALL(hipcub::DeviceRadixSort::SortKeysDescending(
      nullptr, workspace_bytes4, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), num_input_node, 0, sizeof(IdType) * 8,
      cu_stream));
  device->StreamSync(_ctx, stream);

  void *workspace4 = device->AllocWorkspace(_ctx, workspace_bytes4);
  CUDA_CALL(hipcub::DeviceRadixSort::SortKeysDescending(
      workspace4, workspace_bytes4, _node_list, _node_list, num_input_node, 0,
      sizeof(IdType) * 8, cu_stream));
  device->StreamSync(_ctx, stream);

  double step6_time = t6.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 6 finish";

  // 7. get array unique edge number in the order of src nodes.
  //    also count the number of output edges for each nodes.
  //    prefix sum for array of unique edge number.
  Timer t7;
  IdType *num_edge_prefix = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, (num_input_node + 1) * sizeof(IdType)));
  IdType *num_output_prefix = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, (num_input_node + 1) * sizeof(IdType)));
  generate_num_edge<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid0, block0, 0, cu_stream>>>(_node_list, num_input_node, K,
                                        num_edge_prefix, num_output_prefix,
                                        device_table);
  device->StreamSync(_ctx, stream);

  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(workspace2, workspace_bytes2,
                                          num_edge_prefix, num_edge_prefix,
                                          num_input_node + 1, cu_stream));
  device->StreamSync(_ctx, stream);
  double step7_time = t7.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 7 finish";

  // 8. segment-sort the edge for every node using the frequency as key
  //    and the dst as value.
  Timer t8;
  // size_t workspace_bytes5;
  // CUDA_CALL(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
  //     nullptr, workspace_bytes5, static_cast<IdType *>(nullptr),
  //     static_cast<IdType *>(nullptr), static_cast<IdType *>(nullptr),
  //     static_cast<IdType *>(nullptr), _num_unique, num_input_node,
  //     static_cast<IdType *>(nullptr), static_cast<IdType *>(nullptr), 0,
  //     sizeof(IdType) * 8, cu_stream));
  // device->StreamSync(_ctx, stream);

  // void *workspace5 = device->AllocWorkspace(_ctx, workspace_bytes5);
  // CUDA_CALL(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
  //     workspace5, workspace_bytes5, _unique_frequency, _unique_frequency,
  //     _unique_dst, _unique_dst, _num_unique, num_input_node, num_edge_prefix,
  //     num_edge_prefix + 1, 0, sizeof(IdType) * 8, cu_stream));
  // device->StreamSync(_ctx, stream);

  size_t workspace_bytes5;
  Id64Type *key_out = static_cast<Id64Type *>(
      device->AllocDataSpace(_ctx, _unique_list_size * sizeof(Id64Type)));
  IdType *val_out = static_cast<IdType *>(
      device->AllocDataSpace(_ctx, _unique_list_size * sizeof(IdType)));
  CUDA_CALL(hipcub::DeviceRadixSort::SortPairsDescending(
      nullptr, workspace_bytes5, static_cast<Id64Type *>(nullptr),
      static_cast<Id64Type *>(nullptr), static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), _num_unique, 0, sizeof(Id64Type) * 8,
      cu_stream));
  device->StreamSync(_ctx, stream);

  // validate_combine_w_freq<Constant::kCudaBlockSize, Constant::kCudaTileSize>
  //     <<<grid3, block3, 0, cu_stream>>>(_unique_src, _unique_frequency,
  //                                       _unique_combination_key, _num_unique);
  // device->StreamSync(_ctx, stream);
  void *workspace5 = device->AllocWorkspace(_ctx, workspace_bytes5);
  CUDA_CALL(hipcub::DeviceRadixSort::SortPairsDescending(
      workspace5, workspace_bytes5, _unique_combination_key,
      key_out, _unique_dst, val_out, _num_unique, 0,
      sizeof(Id64Type) * 8, cu_stream));
  device->StreamSync(_ctx, stream);
  // validate_combine<Constant::kCudaBlockSize, Constant::kCudaTileSize>
  //     <<<grid3, block3, 0, cu_stream>>>(_unique_src, key_out, _num_unique);
  // device->StreamSync(_ctx, stream);
  device->FreeDataSpace(_ctx, _unique_combination_key);
  device->FreeDataSpace(_ctx, _unique_dst);
  _unique_combination_key = key_out;
  _unique_dst = val_out;
  double step8_time = t8.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 8 finish";

  // 9. prefix the number of output edges for each nodes that we get in step 7
  Timer t9;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(workspace2, workspace_bytes2,
                                          num_output_prefix, num_output_prefix,
                                          num_input_node + 1, cu_stream));
  device->StreamSync(_ctx, stream);
  double step9_time = t9.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 9 finish";

  // 10. copy the result to the output array and set the value of num_output
  /** SXN: bug: unique_frequency is not sorted according to unique_dst*/
  Timer t10;
  compact_output<<<grid2, block2, 0, cu_stream>>>(
      _unique_src, _unique_dst, _unique_frequency, num_input_node, K,
      num_edge_prefix, num_output_prefix, output_src, output_dst, output_data,
      num_output);
  device->StreamSync(_ctx, stream);

  double step10_time = t10.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 10 finish";

  // 11. reset data
  Timer t11;
  reset_node_table<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid0, block0, 0, cu_stream>>>(device_table, _node_list, _num_node);
  Device::Get(_ctx)->StreamSync(_ctx, stream);

  reset_edge_table<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid3, block3, 0, cu_stream>>>(device_table, _unique_node_idx,
                                        _unique_dst, _num_unique);
  Device::Get(_ctx)->StreamSync(_ctx, stream);
  double step11_time = t11.Passed();

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 11 finish";

  _num_node = 0;
  _num_unique = 0;

  device->FreeWorkspace(_ctx, workspace5);
  device->FreeWorkspace(_ctx, num_output_prefix);
  device->FreeWorkspace(_ctx, num_edge_prefix);
  device->FreeWorkspace(_ctx, workspace4);
  device->FreeWorkspace(_ctx, workspace3);
  device->FreeWorkspace(_ctx, unique_idx);
  device->FreeWorkspace(_ctx, tmp_unique_frequency);
  device->FreeWorkspace(_ctx, tmp_unique_dst);
  device->FreeWorkspace(_ctx, tmp_unique_node_idx);
  device->FreeWorkspace(_ctx, device_num_unique);
  device->FreeWorkspace(_ctx, num_unique_prefix);
  device->FreeWorkspace(_ctx, workspace2);
  device->FreeWorkspace(_ctx, workspace1);
  device->FreeWorkspace(_ctx, workspace0);

  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep1Time,
                             step1_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep2Time,
                             step2_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep3Time,
                             step3_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep4Time,
                             step4_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep5Time,
                             step5_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep6Time,
                             step6_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep7Time,
                             step7_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep8Time,
                             step8_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep9Time,
                             step9_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep10Time,
                             step10_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep11Time,
                             step11_time);
}

#else
void FrequencyHashmap::GetTopK(
    IdType *input_src, IdType *input_dst,
    const size_t num_input_edge, const IdType *input_nodes,
    const size_t num_input_node, const size_t K, IdType *output_src,
    IdType *output_dst, IdType *output_data, size_t *num_output,
    StreamHandle stream, uint64_t task_key) {
  const size_t num_tiles0 = RoundUpDiv(num_input_node, Constant::kCudaTileSize);
  const size_t num_tiles1 = RoundUpDiv(num_input_edge, Constant::kCudaTileSize);
  const dim3 grid_input_node(num_tiles0);
  const dim3 grid_input_edge(num_tiles1);

  const dim3 block_input_node(Constant::kCudaBlockSize);
  const dim3 block_input_edge(Constant::kCudaBlockSize);
  dim3 block2(Constant::kCudaBlockSize, 1);
  while (static_cast<size_t>(block2.x) >= 2 * K) {
    block2.x /= 2;
    block2.y *= 2;
  }
  dim3 grid2(RoundUpDiv(num_input_node, static_cast<size_t>(block2.y)));

  auto device_table = MutableDeviceFrequencyHashmap(this);
  auto device = Device::Get(_ctx);
  auto cu_stream = static_cast<hipStream_t>(stream);

  size_t workspace_bytes1;
  size_t workspace_bytes2;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, workspace_bytes1, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), grid_input_edge.x + 1, cu_stream));
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, workspace_bytes2, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), num_input_node + 1, cu_stream));
  device->StreamSync(_ctx, stream);

  void *workspace1 = device->AllocWorkspace(_ctx, workspace_bytes1);
  void *workspace2 = device->AllocWorkspace(_ctx, workspace_bytes2);

  // 1. populate the node table
  Timer t1;
  double step1_time = t1.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 1 finish with "
             << num_input_node << " input nodes with grid " << grid_input_node.x
             << " block " << block_input_node.x;

  // 2. count frequency of every unique edge and
  //    count unique edges for every node
  Timer t2;
  IdType *num_unique_prefix = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, sizeof(IdType) * (grid_input_edge.x + 1)));
  count_frequency_revised<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid_input_edge, block_input_edge, 0, cu_stream>>>(input_src, input_dst, num_input_edge,
                                        _edges_per_node, num_unique_prefix,
                                        device_table);
  device->StreamSync(_ctx, stream);
  /** pos in device_table is now stored in input_src */
  double step2_time = t2.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 2 finish";

  // 3. count the number of unique edges.
  //    prefix sum the the array
  Timer t3;

  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(workspace1, workspace_bytes1,
                                          num_unique_prefix, num_unique_prefix,
                                          grid_input_edge.x + 1, cu_stream));
  device->StreamSync(_ctx, stream);
  double step3_time = t3.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 3 finish";

  // 4. get the array of all unique edges' pos in table
  Timer t4;
  device->CopyDataFromTo(&num_unique_prefix[grid_input_edge.x], 0, &_num_unique, 0,
                         sizeof(IdType), _ctx, CPU(), stream);
  device->StreamSync(_ctx, stream);
  LOG(DEBUG) << "FrequencyHashmap::Before gettopk step 4,  number of unique is " << _num_unique;
  /** location in edge table */
  /** now we reuse input_dst as pos */
  IdType *tmp_unique_pos = input_dst;
  input_dst = nullptr;
  generate_unique_edges_pos<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid_input_edge, block_input_edge, 0, cu_stream>>>(input_src, input_nodes, num_input_node, num_input_edge,
                                        num_unique_prefix, _unique_combination_key, tmp_unique_pos,
                                        _edges_per_node, device_table);
  device->StreamSync(_ctx, stream);
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 4 finish with number of unique "
             << _num_unique;
  double step4_time = t4.Passed();

  // 5. pair-sort unique array using src as key
  //    construct the array of unique dst, unique node idx
  Timer t5;
  /** now we reuse input_src as sort tmp space */
  IdType *alt_val = input_src;
  input_src = nullptr;
  hipcub::DoubleBuffer<Id64Type> keys(_unique_combination_key, reinterpret_cast<Id64Type*>(_unique_node_idx));
  hipcub::DoubleBuffer<IdType>   vals(tmp_unique_pos, alt_val);

  size_t workspace_bytes4;
  CUDA_CALL(hipcub::DeviceRadixSort::SortPairsDescending(
      nullptr, workspace_bytes4, keys, vals, _num_unique, 0, sizeof(Id64Type) * 8,
      cu_stream));
  device->StreamSync(_ctx, stream);

  void *workspace4 = device->AllocWorkspace(_ctx, workspace_bytes4);
  CUDA_CALL(hipcub::DeviceRadixSort::SortPairsDescending(
      workspace4, workspace_bytes4, keys, vals, _num_unique, 0, sizeof(Id64Type) * 8,
      cu_stream));
  device->StreamSync(_ctx, stream);

  _unique_combination_key = keys.Current();
  tmp_unique_pos = vals.Current();
  _unique_node_idx = reinterpret_cast<IdType*>(keys.Alternate());
  _unique_dst = _unique_node_idx + _unique_list_size;
  alt_val = vals.Alternate();

  const size_t num_tiles3 = RoundUpDiv(_num_unique, Constant::kCudaTileSize);
  const dim3 grid_uniq_e(num_tiles3);
  const dim3 block_uniq_e(Constant::kCudaBlockSize);
  construct_unique_edge_list<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid_uniq_e, block_uniq_e, 0, cu_stream>>>(
          _unique_combination_key, tmp_unique_pos,
          _unique_node_idx, _unique_dst, _num_unique, device_table);
  device->StreamSync(_ctx, stream);
  double step5_time = t5.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 5 finish";


  // 6. sort the unique src node array.
  Timer t6;
  double step6_time = t6.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 6 finish";

  // 7. get array unique edge number in the order of src nodes.
  //    also count the number of output edges for each nodes.
  //    prefix sum for array of unique edge number.
  Timer t7;
  IdType *num_edge_prefix = _node_table;
  IdType *num_output_prefix = static_cast<IdType *>(
      device->AllocWorkspace(_ctx, (num_input_node + 1) * sizeof(IdType)));
  generate_num_edge<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid_input_node, block_input_node, 0, cu_stream>>>(input_nodes, num_input_node, K,
                                        num_edge_prefix, num_output_prefix,
                                        device_table);
  device->StreamSync(_ctx, stream);
  /** FIX: only the first num_input_node items are used in num_edge_prefix. reset also only rests these.(notice the grid size) */
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(workspace2, workspace_bytes2,
                                          num_edge_prefix, num_edge_prefix,
                                          num_input_node, cu_stream));
  device->StreamSync(_ctx, stream);

  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(workspace2, workspace_bytes2,
                                          num_output_prefix, num_output_prefix,
                                          num_input_node + 1, cu_stream));
  device->StreamSync(_ctx, stream);
  double step7_time = t7.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 7 finish";

  // 8. nothing to be done
  Timer t8;
  double step8_time = t8.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 8 finish";


  // 9. nothing here
  Timer t9;
  double step9_time = t9.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 9 finish";

  // 10. copy the result to the output array and set the value of num_output
  Timer t10;
  compact_output_revised<<<grid2, block2, 0, cu_stream>>>(
      input_nodes,
      _unique_combination_key, _unique_dst, num_input_node, K,
      num_edge_prefix, num_output_prefix, output_src, output_dst, output_data,
      num_output);
  device->StreamSync(_ctx, stream);

  double step10_time = t10.Passed();
  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 10 finish";

  // 11. reset data
  Timer t11;
  reset_node_table<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid_input_node, block_input_node, 0, cu_stream>>>(device_table, input_nodes, num_input_node);
  Device::Get(_ctx)->StreamSync(_ctx, stream);

  reset_edge_table_revised<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid_uniq_e, block_uniq_e, 0, cu_stream>>>(device_table, tmp_unique_pos,
                                        _unique_dst, _num_unique);
  Device::Get(_ctx)->StreamSync(_ctx, stream);
  double step11_time = t11.Passed();

  LOG(DEBUG) << "FrequencyHashmap::GetTopK step 11 finish";

  _num_unique = 0;

  device->FreeWorkspace(_ctx, num_output_prefix);
  device->FreeWorkspace(_ctx, workspace4);
  device->FreeWorkspace(_ctx, num_unique_prefix);
  device->FreeWorkspace(_ctx, workspace2);
  device->FreeWorkspace(_ctx, workspace1);

  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep1Time,
                             step1_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep2Time,
                             step2_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep3Time,
                             step3_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep4Time,
                             step4_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep5Time,
                             step5_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep6Time,
                             step6_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep7Time,
                             step7_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep8Time,
                             step8_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep9Time,
                             step9_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep10Time,
                             step10_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3RandomWalkTopKStep11Time,
                             step11_time);
}
#endif
}  // namespace cuda
}  // namespace common
}  // namespace samgraph