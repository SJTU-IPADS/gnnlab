#include "hip/hip_runtime.h"
/*
 * Copyright 2022 Institute of Parallel and Distributed Systems, Shanghai Jiao Tong University
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include <hipcub/hipcub.hpp>

#include "cuda_function.h"
#include "cuda_utils.h"
#include "../device.h"
#include "../common.h"
#include "../constant.h"
#include "../logging.h"

namespace samgraph {
namespace common {
namespace cuda {

namespace {

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void count_miss_cache(const IdType *hashtable, const IdType *nodes,
                                 const size_t num_nodes, IdType *miss_counts,
                                 IdType *cache_counts) {
  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = TILE_SIZE * (blockIdx.x + 1);

  using BlockReduce = typename hipcub::BlockReduce<IdType, BLOCK_SIZE>;

  IdType miss_count = 0;
  IdType cache_count = 0;

#pragma unroll
  for (size_t index = threadIdx.x + block_start; index < block_end;
       index += BLOCK_SIZE) {
    if (index < num_nodes) {
      if (hashtable[nodes[index]] == Constant::kEmptyKey) {
        miss_count++;
      } else {
        cache_count++;
      }
    }
  }

  __shared__ typename BlockReduce::TempStorage temp_miss_space;
  __shared__ typename BlockReduce::TempStorage temp_cache_space;

  miss_count = BlockReduce(temp_miss_space).Sum(miss_count);
  cache_count = BlockReduce(temp_cache_space).Sum(cache_count);

  if (threadIdx.x == 0) {
    miss_counts[blockIdx.x] = miss_count;
    cache_counts[blockIdx.x] = cache_count;
    if (blockIdx.x == 0) {
      miss_counts[gridDim.x] = 0;
      cache_counts[gridDim.x] = 0;
    }
  }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void get_miss_index(const IdType *hashtable, const IdType *nodes,
                               const size_t num_nodes,
                               IdType *output_miss_dst_index,
                               IdType *output_miss_src_index,
                               const IdType *miss_counts_prefix) {
  using FlagType = IdType;
  using BlockScan = typename hipcub::BlockScan<FlagType, BLOCK_SIZE>;

  constexpr const IdType VALS_PER_THREAD = TILE_SIZE / BLOCK_SIZE;
  __shared__ typename BlockScan::TempStorage temp_space;
  BlockPrefixCallbackOp<FlagType> prefix_op(0);

  const IdType offset = miss_counts_prefix[blockIdx.x];

  for (IdType i = 0; i < VALS_PER_THREAD; ++i) {
    const IdType index = threadIdx.x + i * BLOCK_SIZE + blockIdx.x * TILE_SIZE;

    FlagType flag;
    if (index < num_nodes && hashtable[nodes[index]] == Constant::kEmptyKey) {
      flag = 1;
    } else {
      flag = 0;
    }

    BlockScan(temp_space).ExclusiveSum(flag, flag, prefix_op);
    __syncthreads();

    if (index < num_nodes && hashtable[nodes[index]] == Constant::kEmptyKey) {
      const IdType pos = offset + flag;
      assert(pos < num_nodes);
      // new node ID in subgraph
      output_miss_dst_index[pos] = index;
      // old node ID in original graph
      output_miss_src_index[pos] = nodes[index];
    }
  }

  // if (threadIdx.x == 0 && blockIdx.x == 0) {
  //   printf("miss count %u, %u\n", miss_counts_prefix[gridDim.x],
  //          miss_counts_prefix[gridDim.x - 1]);
  // }
}

template <size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void get_cache_index(const IdType *hashtable, const IdType *nodes,
                                const size_t num_nodes,
                                IdType *output_cache_dst_index,
                                IdType *output_cache_src_index,
                                const IdType *cache_counts_prefix) {
  using FlagType = IdType;
  using BlockScan = typename hipcub::BlockScan<FlagType, BLOCK_SIZE>;

  constexpr const IdType VALS_PER_THREAD = TILE_SIZE / BLOCK_SIZE;
  __shared__ typename BlockScan::TempStorage temp_space;
  BlockPrefixCallbackOp<FlagType> prefix_op(0);

  const IdType offset = cache_counts_prefix[blockIdx.x];

  for (IdType i = 0; i < VALS_PER_THREAD; ++i) {
    const IdType index = threadIdx.x + i * BLOCK_SIZE + blockIdx.x * TILE_SIZE;

    FlagType flag;
    if (index < num_nodes && hashtable[nodes[index]] != Constant::kEmptyKey) {
      flag = 1;
    } else {
      flag = 0;
    }

    BlockScan(temp_space).ExclusiveSum(flag, flag, prefix_op);
    __syncthreads();

    if (index < num_nodes && hashtable[nodes[index]] != Constant::kEmptyKey) {
      const IdType pos = offset + flag;
      // new node ID in subgraph
      output_cache_dst_index[pos] = index;
      // old node ID in original graph
      output_cache_src_index[pos] = hashtable[nodes[index]];
    }
  }

  // if (threadIdx.x == 0 && blockIdx.x == 0) {
  //   printf("cache count %u, %u\n", cache_counts_prefix[gridDim.x],
  //          cache_counts_prefix[gridDim.x - 1]);
  // }
}

} // namespace

void GetMissCacheIndex(
    IdType *sampler_gpu_hashtable, Context sampler_ctx,
    IdType *output_miss_src_index, IdType *output_miss_dst_index,
    size_t *num_output_miss, IdType *output_cache_src_index,
    IdType *output_cache_dst_index, size_t *num_output_cache,
    const IdType *nodes, const size_t num_nodes, StreamHandle stream) {
  const size_t num_tiles = RoundUpDiv(num_nodes, Constant::kCudaTileSize);
  const dim3 grid(num_tiles);
  const dim3 block(Constant::kCudaBlockSize);

  auto sampler_device = Device::Get(sampler_ctx);
  auto cu_stream = static_cast<hipStream_t>(stream);

  sampler_device->SetDevice(sampler_ctx);

  IdType *miss_prefix_counts =
      static_cast<IdType *>(sampler_device->AllocWorkspace(
          sampler_ctx, sizeof(IdType) * (grid.x + 1)));
  IdType *cache_prefix_counts =
      static_cast<IdType *>(sampler_device->AllocWorkspace(
          sampler_ctx, sizeof(IdType) * (grid.x + 1)));

  // LOG(DEBUG) << "GetMissCacheIndex num nodes " << num_nodes;

  CUDA_CALL(hipSetDevice(sampler_ctx.device_id));
  count_miss_cache<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid, block, 0, cu_stream>>>(sampler_gpu_hashtable, nodes, num_nodes,
                                      miss_prefix_counts, cache_prefix_counts);
  sampler_device->StreamSync(sampler_ctx, stream);

  size_t workspace_bytes;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      nullptr, workspace_bytes, static_cast<IdType *>(nullptr),
      static_cast<IdType *>(nullptr), grid.x + 1, cu_stream));
  sampler_device->StreamSync(sampler_ctx, stream);

  void *workspace =
      sampler_device->AllocWorkspace(sampler_ctx, workspace_bytes);
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      workspace, workspace_bytes, miss_prefix_counts, miss_prefix_counts,
      grid.x + 1, cu_stream));
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      workspace, workspace_bytes, cache_prefix_counts, cache_prefix_counts,
      grid.x + 1, cu_stream));
  sampler_device->StreamSync(sampler_ctx, stream);

  get_miss_index<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid, block, 0, cu_stream>>>(
          sampler_gpu_hashtable, nodes, num_nodes, output_miss_dst_index,
          output_miss_src_index, miss_prefix_counts);
  sampler_device->StreamSync(sampler_ctx, stream);

  get_cache_index<Constant::kCudaBlockSize, Constant::kCudaTileSize>
      <<<grid, block, 0, cu_stream>>>(
          sampler_gpu_hashtable, nodes, num_nodes, output_cache_dst_index,
          output_cache_src_index, cache_prefix_counts);
  sampler_device->StreamSync(sampler_ctx, stream);

  IdType num_miss;
  IdType num_cache;
  sampler_device->CopyDataFromTo(miss_prefix_counts + grid.x, 0, &num_miss, 0,
                                 sizeof(IdType), sampler_ctx, CPU(), stream);
  sampler_device->CopyDataFromTo(cache_prefix_counts + grid.x, 0, &num_cache, 0,
                                 sizeof(IdType), sampler_ctx, CPU(), stream);
  sampler_device->StreamSync(sampler_ctx, stream);

  *num_output_miss = num_miss;
  *num_output_cache = num_cache;

  sampler_device->FreeWorkspace(sampler_ctx, workspace);
  sampler_device->FreeWorkspace(sampler_ctx, cache_prefix_counts);
  sampler_device->FreeWorkspace(sampler_ctx, miss_prefix_counts);
}


}  // namespace cuda
}  // namespace common
}  // namespace samgraph
