#include "hip/hip_runtime.h"
/*
 * Copyright 2022 Institute of Parallel and Distributed Systems, Shanghai Jiao Tong University
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <algorithm>
#include <cassert>
#include <chrono>
#include <cstdio>
#include <hipcub/hipcub.hpp>

#include "../common.h"
#include "../constant.h"
#include "../device.h"
#include "../logging.h"
#include "../profiler.h"
#include "../timer.h"
#include "cuda_function.h"

namespace samgraph {
namespace common {
namespace cuda {

namespace {

__global__ void sample_weighted_khop(
    const IdType *indptr, const IdType *indices, const float *prob_table,
    const IdType *alias_table, const IdType *input, const size_t num_input,
    const size_t fanout, IdType *tmp_src, IdType *tmp_dst,
    hiprandState *random_states, size_t num_random_states) {
  size_t num_task = num_input * fanout;
  size_t thread_id = threadIdx.x + blockDim.x * blockIdx.x;
  size_t task_span = blockDim.x * gridDim.x;

  assert(thread_id < num_random_states);
  // cache the hiprand state
  hiprandState local_state = random_states[thread_id];

  for (size_t task_idx = thread_id; task_idx < num_task;
       task_idx += task_span) {
    const IdType rid = input[task_idx / fanout];
    const IdType off = indptr[rid];
    const IdType len = indptr[rid + 1] - indptr[rid];

    if (len == 0) {
      tmp_src[task_idx] = Constant::kEmptyKey;
    } else {
      tmp_src[task_idx] = rid;
      // choose dst
      size_t k = hiprand(&local_state) % len;
      float r = hiprand_uniform(&local_state);
      if (r < prob_table[off + k]) {
        tmp_dst[task_idx] = indices[off + k];
      } else {
        tmp_dst[task_idx] = alias_table[off + k];
      }
    }
  }
  // restore the state
  random_states[thread_id] = local_state;
}

__global__ void count_edge(IdType *src, IdType *dst, size_t *item_prefix,
                           size_t num_task) {
  size_t thread_id = threadIdx.x + blockDim.x * blockIdx.x;
  size_t task_span = blockDim.x * gridDim.x;

  for (size_t task_idx = thread_id; task_idx < num_task;
       task_idx += task_span) {
    if (task_idx < (num_task - 1)) {
      item_prefix[task_idx] = (src[task_idx] != src[task_idx + 1] ||
                               dst[task_idx] != dst[task_idx + 1]) &&
                              src[task_idx] != Constant::kEmptyKey;
    } else {
      item_prefix[task_idx] = src[task_idx] != Constant::kEmptyKey;
    }
  }

  if (thread_id == 0) {
    item_prefix[num_task] = 0;
  }
}

__global__ void compact_edge(IdType *tmp_src, IdType *tmp_dst, IdType *out_src,
                             IdType *out_dst, size_t *item_prefix,
                             size_t num_task, size_t *num_out) {
  size_t thread_id = threadIdx.x + blockDim.x * blockIdx.x;
  size_t task_span = blockDim.x * gridDim.x;

  for (size_t task_idx = thread_id; task_idx < num_task;
       task_idx += task_span) {
    bool cond;
    if (task_idx < (num_task - 1)) {
      cond = (tmp_src[task_idx] != tmp_src[task_idx + 1] ||
              tmp_dst[task_idx] != tmp_dst[task_idx + 1]) &&
             tmp_src[task_idx] != Constant::kEmptyKey;
    } else {
      cond = tmp_src[task_idx] != Constant::kEmptyKey;
    }

    if (cond) {
      out_src[item_prefix[task_idx]] = tmp_src[task_idx];
      out_dst[item_prefix[task_idx]] = tmp_dst[task_idx];
    }

    // out_src[item_prefix[task_idx]] = tmp_src[task_idx];
    // out_dst[item_prefix[task_idx]] = tmp_dst[task_idx];
  }

  if (thread_id == 0) {
    *num_out = item_prefix[num_task];
  }
}

}  // namespace

void GPUSampleWeightedKHop(const IdType *indptr, const IdType *indices,
                           const float *prob_table, const IdType *alias_table,
                           const IdType *input, const size_t num_input,
                           const size_t fanout, IdType *out_src,
                           IdType *out_dst, size_t *num_out, Context ctx,
                           StreamHandle stream, GPURandomStates *random_states,
                           uint64_t task_key) {
  LOG(DEBUG) << "GPUSample: begin with num_input " << num_input
             << " and fanout " << fanout;
  Timer t0;

  auto sampler_device = Device::Get(ctx);
  auto cu_stream = static_cast<hipStream_t>(stream);
  auto num_sample = num_input * fanout;

  IdType *tmp_src = static_cast<IdType *>(
      sampler_device->AllocWorkspace(ctx, sizeof(IdType) * num_sample));
  IdType *tmp_dst = static_cast<IdType *>(
      sampler_device->AllocWorkspace(ctx, sizeof(IdType) * num_sample));
  LOG(DEBUG) << "GPUSample: cuda tmp_src malloc "
             << ToReadableSize(num_sample * sizeof(IdType));
  LOG(DEBUG) << "GPUSample: cuda tmp_dst malloc "
             << ToReadableSize(num_sample * sizeof(IdType));

  size_t num_threads = Min(num_sample, Constant::kWeightedKHopMaxThreads);
  const dim3 grid(
      RoundUpDiv(num_threads, static_cast<size_t>(Constant::kCudaBlockSize)));
  const dim3 block(Constant::kCudaBlockSize);
  sample_weighted_khop<<<grid, block, 0, cu_stream>>>(
      indptr, indices, prob_table, alias_table, input, num_input, fanout,
      tmp_src, tmp_dst, random_states->GetStates(), random_states->NumStates());
  sampler_device->StreamSync(ctx, stream);

  double sample_time = t0.Passed();
  LOG(DEBUG) << "GPUSample: kernel sampling, time cost: " << sample_time;

  // sort coo
  Timer t1;
  size_t temp_storage_bytes = 0;
  CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(
      nullptr, temp_storage_bytes, tmp_src, tmp_src, tmp_dst, tmp_dst,
      num_sample, 0, sizeof(IdType) * 8, cu_stream));
  sampler_device->StreamSync(ctx, stream);

  void *d_temp_storage =
      sampler_device->AllocWorkspace(ctx, temp_storage_bytes);
  CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(
      d_temp_storage, temp_storage_bytes, tmp_src, tmp_src, tmp_dst, tmp_dst,
      num_sample, 0, sizeof(IdType) * 8, cu_stream));
  sampler_device->StreamSync(ctx, stream);
  sampler_device->FreeWorkspace(ctx, d_temp_storage);
  double sort_coo_time = t1.Passed();
  LOG(DEBUG) << "GPUSample: sort the temporary results, time cost: "
             << sort_coo_time;

  // count the prefix num
  Timer t2;
  size_t *item_prefix = static_cast<size_t *>(
      sampler_device->AllocWorkspace(ctx, sizeof(size_t) * num_sample + 1));
  LOG(DEBUG) << "GPUSample: cuda prefix_num malloc "
             << ToReadableSize(sizeof(int) * num_sample);
  count_edge<<<grid, block, 0, cu_stream>>>(tmp_src, tmp_dst, item_prefix,
                                            num_sample);
  sampler_device->StreamSync(ctx, stream);

  temp_storage_bytes = 0;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(nullptr, temp_storage_bytes,
                                          item_prefix, item_prefix,
                                          num_sample + 1, cu_stream));
  sampler_device->StreamSync(ctx, stream);

  d_temp_storage = sampler_device->AllocWorkspace(ctx, temp_storage_bytes);
  LOG(DEBUG) << "GPUSample: cuda temp_storage for ExclusiveSum malloc "
             << ToReadableSize(temp_storage_bytes);
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                          item_prefix, item_prefix,
                                          num_sample + 1, cu_stream));
  sampler_device->StreamSync(ctx, stream);
  sampler_device->FreeWorkspace(ctx, d_temp_storage);
  double count_edge_time = t2.Passed();
  LOG(DEBUG) << "GPUSample: ExclusiveSum time cost: " << count_edge_time;

  // compact edge
  Timer t3;
  compact_edge<<<grid, block, 0, cu_stream>>>(
      tmp_src, tmp_dst, out_src, out_dst, item_prefix, num_sample, num_out);
  sampler_device->StreamSync(ctx, stream);
  double compact_edge_time = t3.Passed();
  LOG(DEBUG) << "GPUSample: compact_edge time cost: " << compact_edge_time;

  sampler_device->FreeWorkspace(ctx, item_prefix);
  sampler_device->FreeWorkspace(ctx, tmp_src);
  sampler_device->FreeWorkspace(ctx, tmp_dst);

  Profiler::Get().LogStepAdd(task_key, kLogL3KHopSampleCooTime, sample_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3KHopSampleSortCooTime,
                             sort_coo_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3KHopSampleCountEdgeTime,
                             count_edge_time);
  Profiler::Get().LogStepAdd(task_key, kLogL3KHopSampleCompactEdgesTime,
                             compact_edge_time);

  double total_time = t0.Passed();
  LOG(DEBUG) << "GPUSample: succeed total time cost: " << total_time;
}

}  // namespace cuda
}  // namespace common
}  // namespace samgraph