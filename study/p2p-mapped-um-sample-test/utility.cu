#include "hip/hip_runtime.h"
#include <fstream>
#include <cassert>
#include <cstring>
#include <random>
#include "utility.h"

__global__ void delay(volatile int* flag) {
    while(!*flag) {
    }
} 

__global__ void read(int* __restrict__ arr, int len, int* __restrict__ result, int result_len) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t grid_size = blockDim.x * gridDim.x;
    size_t rid = idx % result_len;
#pragma unroll(5)
    for(size_t i = idx; i < len; i += grid_size) {
        result[rid] += arr[i];
    }
}

void perform_sequential_read(
    int grid_size, int block_size, hipStream_t stream, 
    int* arr, int len, int* result, int result_len
) {
    read<<<grid_size, block_size, 0, stream>>>(arr, len, result, result_len);
}

void perform_random_read_int32(
    int grid_size, int block_size, hipStream_t stream,
    int* arr, int len, int* result, int result_len
) {
    std::random_device rd;
    std::mt19937 gen(rd());
    uniform_int_distribution<> dist(0, len - 1);
    read<<<1, 1, 0, stream>>>(arr + dist(gen), 1, result, result_len);
}

tuple<double, double, double> sum_avg_std(const vector<size_t> &vec) {
    double sum = 0;
    double var = 0;
    for(auto x : vec) {
        sum += x;
        var += x * x;
    }
    double avg = sum / vec.size();
    var /= vec.size();
    var -= avg * avg;
    assert(var >= 0);
    return {sum, avg, sqrt(var)};
}

string Dataset::root_path = "/graph-learning/samgraph/";

Dataset::Dataset(string name) : name(name), mm_type(MemoryType::CPU), 
    local_device(hipCpuDeviceId), remote_device(hipCpuDeviceId) {
    string path = root_path + name + "/";
    ifstream indptr(path + "indptr.bin", ios::binary);
    ifstream indices(path + "indices.bin", ios::binary);
    ifstream trainset(path + "train_set.bin", ios::binary);
    if(!indptr) {
        cout << "file " << path + "indptr.bin" << " not exist\n";
        exit(EXIT_FAILURE);
    }
    if(!indices) {
        cout << "file " << path + "indices.bin" << " not exist\n";
        exit(EXIT_FAILURE);
    }
    if(!trainset) {
        cout << "file " << path + "train_set.bin" << " not exist\n";
    }
    indptr.seekg(0, ios::end);
    size_t file_size = indptr.tellg();
    indptr.seekg(0);
    node_num = file_size / 4 - 1;
    // this->indptr = new uint32_t[file_size / 4];
    CUDA_CALL(hipHostAlloc(&this->indptr, file_size, hipHostMallocDefault));
    indptr.read((char*)this->indptr, file_size);
    indptr.close();
    assert(indptr.good());

    indices.seekg(0, ios::end);
    file_size = indices.tellg();
    indices.seekg(0);
    edge_num = file_size / 4;
    // this->indices = new uint32_t[file_size / 4];
    CUDA_CALL(hipHostAlloc(&this->indices, file_size, hipHostMallocDefault));
    indices.read((char*)this->indices, file_size);
    indices.close();
    assert(indices.good());

    trainset.seekg(0, ios::end);
    file_size = trainset.tellg();
    trainset.seekg(0);
    train_num = file_size / 4;
    this->trainset = new uint32_t[train_num];
    trainset.read((char*)this->trainset, file_size);
    trainset.close();
    assert(trainset.good());
}

void Dataset::cpu() {
    if (mm_type == MemoryType::CPU)
        return;
    uint32_t *indptr, *indices;
    // indptr = new uint32_t[node_num + 1];
    // indices = new uint32_t[edge_num];
    CUDA_CALL(hipHostAlloc(&indptr, sizeof(uint32_t) * (node_num + 1), hipHostMallocDefault));
    CUDA_CALL(hipHostAlloc(&indices, sizeof(uint32_t) * (edge_num), hipHostMallocDefault));
    switch (mm_type)
    {
    case MemoryType::HostAllocMapped: 
        CUDA_CALL(hipMemcpy(indptr, this->indptr, sizeof(uint32_t) * (node_num + 1), hipMemcpyDefault));
        CUDA_CALL(hipMemcpy(indices, this->indices, sizeof(uint32_t) * (edge_num), hipMemcpyDefault));
        // memcpy(indptr, this->indptr, sizeof(uint32_t) * (node_num + 1));
        // memcpy(indices, this->indices, sizeof(uint32_t) * (edge_num));
        break;
    case MemoryType::P2P: 
        CUDA_CALL(hipSetDevice(remote_device));
        CUDA_CALL(hipMemcpy(indptr, this->indptr, sizeof(uint32_t) * (node_num + 1), hipMemcpyDefault));
        CUDA_CALL(hipMemcpy(indices, this->indices, sizeof(uint32_t) * (edge_num), hipMemcpyDefault));
        break;
    default :
        assert(false);
    }
    _free_graph();
    this->indptr = indptr;
    this->indices = indices;
    this->mm_type = MemoryType::CPU;
    this->local_device = this->remote_device = hipCpuDeviceId;
}

void Dataset::p2p(int local_device, int remote_device) {
    if (this->mm_type == MemoryType::P2P) {
        return;
    } else if (mm_type != MemoryType::CPU) {
        cpu();
    }
    this->local_device = local_device;
    this->remote_device = remote_device;
    uint32_t *indptr, *indices;

    CUDA_CALL(hipSetDevice(remote_device));
    CUDA_CALL(hipMalloc(&indptr, sizeof(uint32_t) * (node_num + 1)));
    CUDA_CALL(hipMalloc(&indices, sizeof(uint32_t) * edge_num));
    CUDA_CALL(hipMemcpy(indptr, this->indptr, sizeof(uint32_t) * (node_num + 1), hipMemcpyDefault));
    CUDA_CALL(hipMemcpy(indices, this->indices, sizeof(uint32_t) * (edge_num), hipMemcpyDefault));
    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipSetDevice(local_device));
    CUDA_CALL(hipDeviceEnablePeerAccess(remote_device, 0));

    _free_graph();
    this->indptr = indptr;
    this->indices = indices;
    this->mm_type = MemoryType::P2P;
}

void Dataset::hostAllocMapped(int device) {
    if (this->mm_type == MemoryType::HostAllocMapped) {
        return;
    } else if (mm_type != MemoryType::CPU) {
        cpu();
    }
    this->local_device = this->remote_device = device;
    uint32_t *indptr, *indices;

    CUDA_CALL(hipHostAlloc(&indptr, sizeof(uint32_t) * (node_num + 1), hipHostMallocMapped));
    CUDA_CALL(hipHostAlloc(&indices, sizeof(uint32_t) * edge_num, hipHostMallocMapped));
    CUDA_CALL(hipMemcpy(indptr, this->indptr, sizeof(uint32_t) * (node_num + 1), hipMemcpyDefault));
    CUDA_CALL(hipMemcpy(indices, this->indices, sizeof(uint32_t) * (edge_num), hipMemcpyDefault));
    // memcpy(indptr, this->indptr, sizeof(uint32_t) * (node_num + 1));
    // memcpy(indices, this->indices, sizeof(uint32_t) * (edge_num));

    _free_graph();
    this->indptr = indptr;
    this->indices = indices;
    this->mm_type = MemoryType::HostAllocMapped;
}

pair<unique_ptr<uint32_t[]>, unique_ptr<uint32_t[]>> Dataset::get_cpu_graph() {
    auto indptr = make_unique<uint32_t[]>(node_num + 1);
    auto indices = make_unique<uint32_t[]>(edge_num);
    switch (mm_type)
    {
    case MemoryType::CPU:
    case MemoryType::HostAllocMapped:
        CUDA_CALL(hipMemcpy(indptr.get(), this->indptr, sizeof(uint32_t) * (node_num + 1), hipMemcpyDefault));
        CUDA_CALL(hipMemcpy(indices.get(), this->indices, sizeof(uint32_t) * (edge_num), hipMemcpyDefault));
        break;
    case MemoryType::P2P:
        CUDA_CALL(hipSetDevice(remote_device));
        CUDA_CALL(hipMemcpy(indptr.get(), this->indptr, sizeof(uint32_t) * (node_num + 1), hipMemcpyDefault));
        CUDA_CALL(hipMemcpy(indices.get(), this->indices, sizeof(uint32_t) * edge_num, hipMemcpyDefault));
        break;
    case MemoryType::UM_CUDA_CPU:
    case MemoryType::UM_CUDA_CUDA:
    default:
        assert(false);
    }
    return {move(indptr), move(indices)};
}

Dataset::~Dataset() {
    _free_graph();
    delete[] this->trainset;
}

void Dataset::_free_graph() {
    switch (mm_type)
    {
    case MemoryType::CPU:
        // delete[] this->indptr;
        // delete[] this->indices;
        CUDA_CALL(hipHostFree(this->indptr));
        CUDA_CALL(hipHostFree(this->indices));
        return ;
    case MemoryType::HostAllocMapped:
        CUDA_CALL(hipHostFree(this->indptr));
        CUDA_CALL(hipHostFree(this->indices));
        return;
    case MemoryType::P2P:
        CUDA_CALL(hipSetDevice(local_device));
        CUDA_CALL(hipDeviceDisablePeerAccess(remote_device));
        CUDA_CALL(hipSetDevice(remote_device));
        CUDA_CALL(hipFree(this->indptr));
        CUDA_CALL(hipFree(this->indices));
        return;
    default:
        assert(false);
    }
    
}