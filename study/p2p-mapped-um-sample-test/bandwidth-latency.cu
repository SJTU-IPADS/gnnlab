#include "hip/hip_runtime.h"
#include <bits/stdc++.h>

#include "utility.h"
using namespace std;

struct Array
{
    int* data;
    size_t size;
};

class MemoryTestCase {
public:
    string name;
    int device;
    Array read_buf;
    Array read_result;
    hipStream_t stream;
    MemoryTestCase(string name, int device, size_t read_buf_size, size_t read_result_size = 10) 
        : device(device), name(name), 
        read_buf_size(read_buf_size), read_result_size(read_result_size), 
        finished(false), time(-1) {}
    virtual ~MemoryTestCase() {}

    virtual void Init() {
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        CUDA_CALL(hipMalloc(&read_result.data, read_result_size * sizeof(int)));
        read_result.size = read_result_size;
    }
    virtual void Clear(float time) {
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipStreamDestroy(stream));
        CUDA_CALL(hipFree(read_result.data));
        read_result.data = nullptr;
        read_result.size = 0;
        finished = true;
        this->time = time;
    }

    size_t ReadSize() const {
        if (!finished) {
            cout << "Warning: " << name << " Get ReadSize before test finish\n";
        }
        return sizeof(int) * read_buf_size;
    }
    double Time() const {
        if (!finished) {
            cout << "Warning: " << name << " Get time before test finish\n";
        }
        return time;
    }
protected:
    bool finished;
    const size_t read_buf_size;
    const size_t read_result_size;
    double time; // ms
};

class Local : public MemoryTestCase {
public:
    Local(int device, size_t elem_num)
        : MemoryTestCase("Local", device, elem_num) {}

    virtual void Init() override {
        MemoryTestCase::Init();
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipMalloc(&read_buf.data, read_buf_size * sizeof(int)));
        read_buf.size = read_buf_size;
        CUDA_CALL(hipMemset(read_buf.data, 0x0f, sizeof(int) * read_buf.size));
    }
    virtual void Clear(float time) override {
        MemoryTestCase::Clear(time);
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipFree(read_buf.data));
        read_buf.data = nullptr;
        read_buf.size = 0;
    }
};

class P2P : public MemoryTestCase {
public:
    int remote_device;
    P2P(int local_device, int remote_device, size_t elem_num) 
        : MemoryTestCase("P2P", local_device, elem_num), remote_device(remote_device) {
    }
    
    virtual void Init() override {
        MemoryTestCase::Init();
        CUDA_CALL(hipSetDevice(remote_device));
        CUDA_CALL(hipMalloc(&read_buf.data, read_buf_size * sizeof(int)));
        read_buf.size = read_buf_size;
        CUDA_CALL(hipMemset(read_buf.data, 0x0f, sizeof(int) * read_buf.size));
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipDeviceEnablePeerAccess(remote_device, 0));
    }
    virtual void Clear(float time) override {
        MemoryTestCase::Clear(time);
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipDeviceDisablePeerAccess(remote_device));
        CUDA_CALL(hipSetDevice(remote_device));
        CUDA_CALL(hipFree(read_buf.data));
        read_buf.data = nullptr;
        read_buf.size = 0;
    }
};

class HostMapped : public MemoryTestCase {
public:
    HostMapped(int device, size_t elem_num)
        : MemoryTestCase("hipHostMallocMapped", device, elem_num) {}
    
    virtual void Init() override {
        MemoryTestCase::Init();
        CUDA_CALL(hipHostAlloc(&read_buf.data, sizeof(int) * read_buf_size, hipHostMallocMapped));
        read_buf.size = read_buf_size;
        memset(read_buf.data, 0x0f, sizeof(int) * read_buf.size);
    }
    virtual void Clear(float time) override {
        MemoryTestCase::Clear(time);
        CUDA_CALL(hipHostFree(read_buf.data));
        read_buf.data = nullptr;
        read_buf.size = 0;
    }
};

class UM_CUDA_CUDA : public MemoryTestCase {
public:
    int remote_device;
    UM_CUDA_CUDA(int local_device, int remote_device, size_t elem_num)
        : MemoryTestCase("UM-cuda+cuda", local_device, elem_num), remote_device(remote_device) {}

    virtual void Init() override {
        MemoryTestCase::Init();
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipMallocManaged(&read_buf.data, sizeof(int) * read_buf_size));
        read_buf.size = read_buf_size;
        CUDA_CALL(hipMemAdvise(read_buf.data, sizeof(int) * read_buf.size, 
            hipMemAdviseSetPreferredLocation, remote_device));
        CUDA_CALL(hipMemAdvise(read_buf.data, sizeof(int) * read_buf.size,
            hipMemAdviseSetAccessedBy, device));

        auto ptr = make_unique<int[]>(read_buf.size);
        memset(ptr.get(), 0x0f, sizeof(int) * read_buf.size);
        CUDA_CALL(hipMemcpy(read_buf.data, ptr.get(), sizeof(int) * read_buf.size, hipMemcpyHostToDevice));
    }
    virtual void Clear(float time) override {
        MemoryTestCase::Clear(time);
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipFree(read_buf.data));
        read_buf.data = nullptr;
        read_buf.size = 0;
    }
};

class UM_CUDA_CPU : public MemoryTestCase {
public:
    UM_CUDA_CPU(int device, size_t elem_num)
        : MemoryTestCase("UM-cuda+cpu", device, elem_num) {}
    
    virtual void Init() override {
        MemoryTestCase::Init();
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipMallocManaged(&read_buf.data, sizeof(int) * read_buf_size));
        read_buf.size = read_buf_size;
        CUDA_CALL(hipMemAdvise(read_buf.data, sizeof(int) * read_buf.size,
            hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
        CUDA_CALL(hipMemAdvise(read_buf.data, sizeof(int) * read_buf.size,
            hipMemAdviseSetAccessedBy, device));

        auto ptr = make_unique<int[]>(read_buf.size);
        memset(ptr.get(), 0x0f, sizeof(int) * read_buf.size);
        CUDA_CALL(hipMemcpy(read_buf.data, ptr.get(), sizeof(int) * read_buf_size, hipMemcpyHostToDevice));
    }
    virtual void Clear(float time) override {
        MemoryTestCase::Clear(time);
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipFree(read_buf.data));
        read_buf.data = nullptr;
        read_buf.size = 0;
    }
}; 

template<size_t elem_num, auto perform_read, decltype(perform_read) overhead = nullptr>
class SMReadTest {
public:
    SMReadTest(int local_device, int remote_deivce, int repeat = 5) 
    : repeat(repeat) {
        // check p2p is enble, which will have effect on um-cuda+cuda and p2p test case
        int access = 0;
        CUDA_CALL(hipDeviceCanAccessPeer(&access, local_device, remote_deivce));
        if (!access) {
            cout << "device " << local_device << " " << remote_deivce
                 << " do not support p2p access, abort testing\n";
        }
        env.push_back(make_unique<Local>(local_device, elem_num));
        env.push_back(make_unique<HostMapped>(local_device, elem_num));
        env.push_back(make_unique<P2P>(local_device, remote_deivce, elem_num));
        env.push_back(make_unique<UM_CUDA_CUDA>(local_device, remote_deivce, elem_num));
        env.push_back(make_unique<UM_CUDA_CPU>(local_device, elem_num));
    }
    void Run() {
        cout << __LINE__ << "\n";
        volatile int* start_flag;
        hipHostAlloc(&start_flag, sizeof(int), hipHostMallocPortable);
        vector<hipEvent_t> start(env.size()), end(env.size());
        vector<hipEvent_t> start_oh(env.size()), end_oh(env.size());
        for(int i = 0; i < env.size(); i++) {
            // cout << env[i]->name << " " << env[i]->device << "\n";
            CUDA_CALL(hipSetDevice(env[i]->device));
            CUDA_CALL(hipEventCreate(&start[i]));
            CUDA_CALL(hipEventCreate(&end[i]));
            CUDA_CALL(hipEventCreate(&start_oh[i]));
            CUDA_CALL(hipEventCreate(&end_oh[i]));
        }
        int block_size = 0, grid_size = 0;
        CUDA_CALL(hipOccupancyMaxPotentialBlockSize(&block_size, &grid_size, read));

        auto measure_kernel = [&](
            MemoryTestCase* env, hipEvent_t start, hipEvent_t end, decltype(perform_read) kernel
        ) -> float {
            *start_flag = 0;
            CUDA_CALL(hipSetDevice(env->device));
            CUDA_CALL(hipStreamSynchronize(env->stream));
            delay<<<1, 1, 0, env->stream>>>(start_flag);
            CUDA_CALL(hipEventRecord(start, env->stream));
            for(int r = 0; r < this->repeat; r++) {
                kernel(grid_size, block_size, env->stream, 
                    env->read_buf.data, env->read_buf.size, 
                    env->read_result.data, env->read_result.size);
            }
            CUDA_CALL(hipEventRecord(end, env->stream));

            *start_flag = 1;
            CUDA_CALL(hipStreamSynchronize(env->stream));
            float ms;
            CUDA_CALL(hipEventElapsedTime(&ms, start, end));
            return ms;
        };

        for(int i = 0; i < env.size(); i++) {
            report_process(i);
            env[i]->Init();
            float kernel_ms = measure_kernel(env[i].get(), start[i], end[i], perform_read);
            if (overhead != nullptr) {
                float overhead_ms = measure_kernel(env[i].get(), start_oh[i], end_oh[i], overhead);
                // cout << "\n" << env[i]->name << " overhead_ms " << overhead_ms << " kernel_ms " << kernel_ms << "\n";
                kernel_ms -= overhead_ms;
            }
            env[i]->Clear(kernel_ms);
        }
        report_process(env.size());
        hipFree((void*)start_flag);
        for(int i = 0; i < env.size(); i++) {
            CUDA_CALL(hipSetDevice(env[i]->device));
            CUDA_CALL(hipEventDestroy(start[i]));
            CUDA_CALL(hipEventDestroy(end[i]));
            CUDA_CALL(hipEventDestroy(start_oh[i]));
            CUDA_CALL(hipEventDestroy(end_oh[i]))
        }
    }
    virtual void Statistic() = 0;
protected:
    vector<unique_ptr<MemoryTestCase>> env;
    int repeat;
    void report_process(int i) {
        int done = 10.0 * i / env.size();
        int todo = 10 - done;
        cout << "\rSMTesetRead<elem_num=" << elem_num << ">: ";
        cout << "[";
        cout << string(done, '#') << string(todo, '.') << "] ";
        cout << i << "/" << env.size() << std::flush;
        if(i == env.size()) {
            cout << "\n";
        }
    }
};

template<size_t elem_num = (1ULL << 28)>
class BandWitdhTest : public SMReadTest<elem_num, perform_sequential_read> {
public:
    BandWitdhTest(int local_device, int remote_device, int repeat = 5)
        : SMReadTest<elem_num, perform_sequential_read>(local_device, remote_device, repeat) {}
    virtual void Statistic() override {
        cout << "Sequential Thpt Test Result:\n";
        cout << "----------------------\n";
        cout.setf(ios::left, ios::adjustfield);
        size_t fwid1 = string{"MemoryType"}.size();
        for(auto &e : this->env) {
            fwid1 = std::max(fwid1, e->name.size());
        }
        size_t fwid2 = string{"Throughput(GB/s)"}.size();
        fwid1 += 4;
        fwid2 += 4;
        cout.width(fwid1); cout << "MemoryType" << "|  ";
        cout.width(fwid2); cout << "Throughput(GB/s)" << "\n";
        for(auto &e : this->env) {
            auto time = e->Time();
            auto bandwidth = 1.0 * e->ReadSize() / 1024 / 1024 / 1024 / time * 1000; // gb/s
            bandwidth *= this->repeat;
            cout.width(fwid1); cout << e->name << "|  ";
            cout.width(fwid2); cout << bandwidth << "\n";
        }
        cout << "\n";
    }
};

template<size_t elem_num = (1ULL << 28)>
class LatencyTest : public SMReadTest<elem_num, perform_random_read_int32> {
public: 
    LatencyTest(int local_device, int remote_device, int repeat = 5)
        : SMReadTest<elem_num, perform_random_read_int32>(local_device, remote_device, repeat) {};
    virtual void Statistic() override {
        cout << "Latency Test Result:\n";
        cout << "--------------------\n";
        cout.setf(ios::left, ios::adjustfield);
        size_t fwid1 = string{"MemoryType"}.size();
        for(auto &e : this->env) {
            fwid1 = std::max(fwid1, e->name.size());
        }
        size_t fwid2 = string{"Latency(us)"}.size();
        fwid1 += 4;
        fwid2 += 4;
        cout.width(fwid1); cout << "MemoryType" << "|  ";
        cout.width(fwid2); cout << "Latency(us)" << "\n";
        for(auto &e : this->env) {
            auto time = e->Time();
            auto latency = time * 1e3; // us
            latency /= this->repeat;
            cout.width(fwid1);
            cout << e->name << "|  ";
            cout.width(fwid2);
            cout << latency << "\n";
            // cout << "\t" << e->name << "\t|"
            //      << "\t" << latency << "\t\n";
        }
        cout << "\n";
    }
};

template<size_t elem_num = (1ULL << 28)>
class RandomBandwidth : public SMReadTest<elem_num, perform_random_read, perform_random_read_overhead> {
public:
    RandomBandwidth(int local_device, int remote_device, int repeat = 5)
        : SMReadTest<elem_num, perform_random_read, perform_random_read_overhead>(local_device, remote_device, repeat) {}
    virtual void Statistic() override {
        cout << "Random Thpt Test Result:\n";
        cout << "----------------------------\n";
        cout.setf(ios::left, ios::adjustfield);
        string l = "MemoryType", r = "Throughput(GB/s)";
        size_t fw1 = l.size();
        size_t fw2 = r.size();
        for(auto &e : this->env) {
            fw1 = std::max(fw1, e->name.size());
        }
        fw1 += 4, fw2 += 4;
        cout.width(fw1); cout << l << "| ";
        cout.widen(fw2); cout << r << "\n";
        for(auto &e : this->env) {
            auto time = e->Time();
            auto bandwidth = 1.0 * e->ReadSize() / 1024 / 1024 / 1024 / time * 1000; // gb/s
            bandwidth *= this->repeat;
            cout.width(fw1); cout << e->name << "|  ";
            cout.width(fw2); cout << bandwidth << "\n";
        }
        cout << "\n";
    }
};

template<auto perform_kerenl, size_t elem_num = (1ULL << 28), decltype(perform_kerenl) overhead = nullptr> 
class KernelTimeTest : public SMReadTest<elem_num, perform_kerenl, overhead> {
public:
    KernelTimeTest(int local_device, int remote_device, int repeat = 5) 
        : SMReadTest<elem_num, perform_kerenl, overhead>(local_device, remote_device, repeat) {}
    virtual void Statistic() override {
        std::string title = std::string(test_name(perform_kerenl)) + " Time Test:\n";
        cout << string(title.size(), '-') << "\n";
        cout.setf(ios::left, ios::adjustfield);
        string l = "MemoryType", r = "Time(ms)";
        size_t fw1 = l.size();
        size_t fw2 = r.size();
        for (auto &e : this->env) {
            fw1 = std::max(fw1, e->name.size());
        }
        fw1 += 4, fw2 += 4;
        cout.width(fw1); cout << l << "| ";
        cout.width(fw2); cout << r << "\n";
        for (auto &e: this->env) {
            auto time = e->Time();
            time /= this->repeat;
            cout.width(fw1); cout << e->name << "| ";
            cout.width(fw2); cout << time << "\n";
        }
        cout << "\n";
    }
};

template<size_t page_size>
using RoffRlookbehindTime = KernelTimeTest<perform_random_off_random_lookbehind<page_size>>;

int main() {
    cout << __LINE__ << "\n";

    BandWitdhTest bandwidth_test(0, 1, 1);
    bandwidth_test.Run();
    bandwidth_test.Statistic();

    LatencyTest latency_test(0, 1, 100);
    latency_test.Run();
    latency_test.Statistic();

    RandomBandwidth random_bandwidth_test(0, 1, 1);
    random_bandwidth_test.Run();
    random_bandwidth_test.Statistic();

    RoffRlookbehindTime<4096> random_off_random_lookbehind_test(0, 1, 1);
    random_off_random_lookbehind_test.Run();
    random_off_random_lookbehind_test.Statistic();

    // RandomDivergenceTime random_divergence_time_test(0, 1, 1);
    // random_divergence_time_test.Run();
    // random_divergence_time_test.Statistic();
}