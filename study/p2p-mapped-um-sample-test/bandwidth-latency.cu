#include "hip/hip_runtime.h"
#include <bits/stdc++.h>

#include "utility.h"
using namespace std;

struct Array
{
    int* data;
    size_t size;
};

class MemoryTestCase {
public:
    string name;
    int device;
    Array read_buf;
    Array read_result;
    hipStream_t stream;
    MemoryTestCase(string name, int device, size_t read_buf_size, size_t read_result_size = 10) 
        : device(device), name(name), 
        read_buf_size(read_buf_size), read_result_size(read_result_size), 
        finished(false), time(-1) {}
    virtual ~MemoryTestCase() {}

    virtual void Init() {
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        CUDA_CALL(hipMalloc(&read_result.data, read_result_size * sizeof(int)));
        read_result.size = read_result_size;
    }
    virtual void Clear(float time) {
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipStreamDestroy(stream));
        CUDA_CALL(hipFree(read_result.data));
        read_result.data = nullptr;
        read_result.size = 0;
        finished = true;
        this->time = time;
    }

    size_t ReadSize() const {
        if (!finished) {
            cout << "Warning: " << name << " Get ReadSize before test finish\n";
        }
        return sizeof(int) * read_buf_size;
    }
    double Time() const {
        if (!finished) {
            cout << "Warning: " << name << " Get time before test finish\n";
        }
        return time;
    }
protected:
    bool finished;
    const size_t read_buf_size;
    const size_t read_result_size;
    double time; // ms
};

class Local : public MemoryTestCase {
public:
    Local(int device, size_t elem_num)
        : MemoryTestCase("Local", device, elem_num) {}

    virtual void Init() override {
        MemoryTestCase::Init();
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipMalloc(&read_buf.data, read_buf_size * sizeof(int)));
        read_buf.size = read_buf_size;
        CUDA_CALL(hipMemset(read_buf.data, 0x0f, sizeof(int) * read_buf.size));
    }
    virtual void Clear(float time) override {
        MemoryTestCase::Clear(time);
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipFree(read_buf.data));
        read_buf.data = nullptr;
        read_buf.size = 0;
    }
};

class P2P : public MemoryTestCase {
public:
    int remote_device;
    P2P(int local_device, int remote_device, size_t elem_num) 
        : MemoryTestCase("P2P", local_device, elem_num), remote_device(remote_device) {
    }
    
    virtual void Init() override {
        MemoryTestCase::Init();
        CUDA_CALL(hipSetDevice(remote_device));
        CUDA_CALL(hipMalloc(&read_buf.data, read_buf_size * sizeof(int)));
        read_buf.size = read_buf_size;
        CUDA_CALL(hipMemset(read_buf.data, 0x0f, sizeof(int) * read_buf.size));
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipDeviceEnablePeerAccess(remote_device, 0));
    }
    virtual void Clear(float time) override {
        MemoryTestCase::Clear(time);
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipDeviceDisablePeerAccess(remote_device));
        CUDA_CALL(hipSetDevice(remote_device));
        CUDA_CALL(hipFree(read_buf.data));
        read_buf.data = nullptr;
        read_buf.size = 0;
    }
};

class HostMapped : public MemoryTestCase {
public:
    HostMapped(int device, size_t elem_num)
        : MemoryTestCase("hipHostMallocMapped", device, elem_num) {}
    
    virtual void Init() override {
        MemoryTestCase::Init();
        CUDA_CALL(hipHostAlloc(&read_buf.data, sizeof(int) * read_buf_size, hipHostMallocMapped));
        read_buf.size = read_buf_size;
        memset(read_buf.data, 0x0f, sizeof(int) * read_buf.size);
    }
    virtual void Clear(float time) override {
        MemoryTestCase::Clear(time);
        CUDA_CALL(hipHostFree(read_buf.data));
        read_buf.data = nullptr;
        read_buf.size = 0;
    }
};

class UM_CUDA_CUDA : public MemoryTestCase {
public:
    int remote_device;
    UM_CUDA_CUDA(int local_device, int remote_device, size_t elem_num)
        : MemoryTestCase("UM-cuda+cuda", local_device, elem_num), remote_device(remote_device) {}

    virtual void Init() override {
        MemoryTestCase::Init();
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipMallocManaged(&read_buf.data, sizeof(int) * read_buf_size));
        read_buf.size = read_buf_size;
        CUDA_CALL(hipMemAdvise(read_buf.data, sizeof(int) * read_buf.size, 
            hipMemAdviseSetPreferredLocation, remote_device));
        CUDA_CALL(hipMemAdvise(read_buf.data, sizeof(int) * read_buf.size,
            hipMemAdviseSetAccessedBy, device));

        auto ptr = make_unique<int[]>(read_buf.size);
        memset(ptr.get(), 0x0f, sizeof(int) * read_buf.size);
        CUDA_CALL(hipMemcpy(read_buf.data, ptr.get(), sizeof(int) * read_buf.size, hipMemcpyHostToDevice));
    }
    virtual void Clear(float time) override {
        MemoryTestCase::Clear(time);
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipFree(read_buf.data));
        read_buf.data = nullptr;
        read_buf.size = 0;
    }
};

class UM_CUDA_CPU : public MemoryTestCase {
public:
    UM_CUDA_CPU(int device, size_t elem_num)
        : MemoryTestCase("UM-cuda+cpu", device, elem_num) {}
    
    virtual void Init() override {
        MemoryTestCase::Init();
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipMallocManaged(&read_buf.data, sizeof(int) * read_buf_size));
        read_buf.size = read_buf_size;
        CUDA_CALL(hipMemAdvise(read_buf.data, sizeof(int) * read_buf.size,
            hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
        CUDA_CALL(hipMemAdvise(read_buf.data, sizeof(int) * read_buf.size,
            hipMemAdviseSetAccessedBy, device));

        auto ptr = make_unique<int[]>(read_buf.size);
        memset(ptr.get(), 0x0f, sizeof(int) * read_buf.size);
        CUDA_CALL(hipMemcpy(read_buf.data, ptr.get(), sizeof(int) * read_buf_size, hipMemcpyHostToDevice));
    }
    virtual void Clear(float time) override {
        MemoryTestCase::Clear(time);
        CUDA_CALL(hipSetDevice(device));
        CUDA_CALL(hipFree(read_buf.data));
        read_buf.data = nullptr;
        read_buf.size = 0;
    }
}; 

template<size_t elem_num, auto perform_read>
class SMReadTest {
public:
    SMReadTest(int local_device, int remote_deivce, int repeat = 5) 
    : repeat(repeat) {
        // check p2p is enble, which will have effect on um-cuda+cuda and p2p test case
        int access = 0;
        CUDA_CALL(hipDeviceCanAccessPeer(&access, local_device, remote_deivce));
        if (!access) {
            cout << "device " << local_device << " " << remote_deivce
                 << " do not support p2p access, abort testing\n";
        }
        env.push_back(make_unique<Local>(local_device, elem_num));
        env.push_back(make_unique<HostMapped>(local_device, elem_num));
        env.push_back(make_unique<P2P>(local_device, remote_deivce, elem_num));
        env.push_back(make_unique<UM_CUDA_CUDA>(local_device, remote_deivce, elem_num));
        env.push_back(make_unique<UM_CUDA_CPU>(local_device, elem_num));
    }
    void Run() {
        volatile int* start_flag;
        hipHostAlloc(&start_flag, sizeof(int), hipHostMallocPortable);
        vector<hipEvent_t> start(env.size()), end(env.size());
        for(int i = 0; i < env.size(); i++) {
            // cout << env[i]->name << " " << env[i]->device << "\n";
            CUDA_CALL(hipSetDevice(env[i]->device));
            CUDA_CALL(hipEventCreate(&start[i]));
            CUDA_CALL(hipEventCreate(&end[i]));
        }
        int block_size = 0, grid_size = 0;
        CUDA_CALL(hipOccupancyMaxPotentialBlockSize(&block_size, &grid_size, read));
        for(int i = 0; i < env.size(); i++) {
            report_process(i);
            *start_flag = 0;
            env[i]->Init();
            CUDA_CALL(hipSetDevice(env[i]->device));
            CUDA_CALL(hipStreamSynchronize(env[i]->stream));
            delay<<<1, 1, 0, env[i]->stream>>>(start_flag);
            CUDA_CALL(hipEventRecord(start[i], env[i]->stream));
            for(int r = 0; r < repeat; r++) {
                // read<<<grid_size, block_size, 0, env[i]->stream>>>(
                //     env[i]->read_buf.data, env[i]->read_buf.size, 
                //     env[i]->read_result.data, env[i]->read_result.size);
                perform_read(grid_size, block_size, env[i]->stream, 
                    env[i]->read_buf.data, env[i]->read_buf.size, 
                    env[i]->read_result.data, env[i]->read_result.size);
            }
            CUDA_CALL(hipEventRecord(end[i], env[i]->stream));
            
            *start_flag = 1;
            CUDA_CALL(hipStreamSynchronize(env[i]->stream));
            float ms;
            CUDA_CALL(hipEventElapsedTime(&ms, start[i], end[i]));
            env[i]->Clear(ms);
        }
        report_process(env.size());
        hipFree((void*)start_flag);
        for(int i = 0; i < env.size(); i++) {
            CUDA_CALL(hipSetDevice(env[i]->device));
            CUDA_CALL(hipEventDestroy(start[i]));
            CUDA_CALL(hipEventDestroy(end[i]));
        }
    }
    virtual void Statistic() = 0;
protected:
    vector<unique_ptr<MemoryTestCase>> env;
    int repeat;
    void report_process(int i) {
        int done = 10.0 * i / env.size();
        int todo = 10 - done;
        cout << "\rSMTesetRead<elem_num=" << elem_num << ">: ";
        cout << "[";
        cout << string(done, '#') << string(todo, '.') << "] ";
        cout << i << "/" << env.size() << std::flush;
        if(i == env.size()) {
            cout << "\n";
        }
    }
};

template<size_t elem_num = (1ULL << 28)>
class BandWitdhTest : public SMReadTest<elem_num, perform_sequential_read> {
public:
    BandWitdhTest(int local_device, int remote_device, int repeat = 5)
        : SMReadTest<elem_num, perform_sequential_read>(local_device, remote_device, repeat) {}
    virtual void Statistic() override {
        cout << "BandWidth Test Result:\n";
        cout << "----------------------\n";
        cout.setf(ios::left, ios::adjustfield);
        size_t fwid1 = string{"MemoryType"}.size();
        for(auto &e : this->env) {
            fwid1 = std::max(fwid1, e->name.size());
        }
        size_t fwid2 = string{"BandWidth(GB/s)"}.size();
        fwid1 += 4;
        fwid2 += 4;
        cout.width(fwid1); cout << "MemoryType" << "|  ";
        cout.width(fwid2); cout << "BandWidth(GB/s)" << "\n";
        for(auto &e : this->env) {
            auto time = e->Time();
            auto bandwidth = 1.0 * e->ReadSize() / 1024 / 1024 / 1024 / time * 1000; // gb/s
            bandwidth *= this->repeat;
            cout.width(fwid1); cout << e->name << "|  ";
            cout.width(fwid2); cout << bandwidth << "\n";
        }
        cout << "\n";
    }
};

template<size_t elem_num = (1ULL << 28)>
class LatencyTest : public SMReadTest<elem_num, perform_random_read_int32> {
public: 
    LatencyTest(int local_device, int remote_device, int repeat = 5)
        : SMReadTest<elem_num, perform_random_read_int32>(local_device, remote_device, repeat) {};
    virtual void Statistic() override {
        cout << "Latency Test Result:\n";
        cout << "--------------------\n";
        cout.setf(ios::left, ios::adjustfield);
        size_t fwid1 = string{"MemoryType"}.size();
        for(auto &e : this->env) {
            fwid1 = std::max(fwid1, e->name.size());
        }
        size_t fwid2 = string{"Latency(us)"}.size();
        fwid1 += 4;
        fwid2 += 4;
        cout.width(fwid1); cout << "MemoryType" << "|  ";
        cout.width(fwid2); cout << "Latency(us)" << "\n";
        for(auto &e : this->env) {
            auto time = e->Time();
            auto latency = time * 1e3; // us
            latency /= this->repeat;
            cout.width(fwid1);
            cout << e->name << "|  ";
            cout.width(fwid2);
            cout << latency << "\n";
            // cout << "\t" << e->name << "\t|"
            //      << "\t" << latency << "\t\n";
        }
        cout << "\n";
    }
};

template<size_t elem_num = (1ULL << 28)>
class RandomBandwidth : public SMReadTest<elem_num, perform_random_read> {
public:
    RandomBandwidth(int local_device, int remote_device, int repeat = 5)
        : SMReadTest<elem_num, perform_random_read>(local_device, remote_device, repeat) {}
    virtual void Statistic() override {
        cout << "RandomBandwidth Test Result:\n";
        cout << "----------------------------\n";
        cout.setf(ios::left, ios::adjustfield);
        string l = "MemoryType", r = "RandomBandwidth(GB/s)";
        size_t fw1 = l.size();
        size_t fw2 = r.size();
        for(auto &e : this->env) {
            fw1 = std::max(fw1, e->name.size());
        }
        fw1 += 4, fw2 += 4;
        cout.width(fw1); cout << l << "| ";
        cout.widen(fw2); cout << r << "\n";
        for(auto &e : this->env) {
            auto time = e->Time();
            auto bandwidth = 1.0 * e->ReadSize() / 1024 / 1024 / 1024 / time * 1000; // gb/s
            bandwidth *= this->repeat;
            cout.width(fw1); cout << e->name << "|  ";
            cout.width(fw2); cout << bandwidth << "\n";
        }
        cout << "\n";
    }
};

int main() {
    BandWitdhTest bandwidth_test(0, 1, 1);
    bandwidth_test.Run();
    bandwidth_test.Statistic();

    LatencyTest latency_test(0, 1, 100);
    latency_test.Run();
    latency_test.Statistic();

    RandomBandwidth random_bandwidth_test(0, 1, 1);
    random_bandwidth_test.Run();
    random_bandwidth_test.Statistic();
}