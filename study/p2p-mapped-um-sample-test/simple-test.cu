#include <bits/stdc++.h>
#include "utility.h"
using namespace std;

int main() {
    hipStream_t stream;
    hipEvent_t start, end;
    volatile int* flag;

    int *arr, *result;
    int len = 1;

    CUDA_CALL(hipSetDevice(0));
    CUDA_CALL(hipMalloc(&arr, sizeof(int) * len));
    CUDA_CALL(hipMalloc(&result, sizeof(int) * len));
    CUDA_CALL(hipMemset(arr, 0, sizeof(int) * len));
    CUDA_CALL(hipMemset(result, 0, sizeof(int) * len));
    CUDA_CALL(hipHostAlloc(&flag, sizeof(int), hipHostMallocPortable));
    CUDA_CALL(hipStreamCreate(&stream));
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&end));

    int r = 100;

    CUDA_CALL(hipStreamSynchronize(stream));
    *flag = 0;
    delay<<<1, 1, 0, stream>>>(flag);
    CUDA_CALL(hipEventRecord(start, stream));
    for(int i = 0; i < r; i++) {
        arr += 0;
        read<<<1, 1, 0, stream>>>(arr, len, result, len);
    }
    CUDA_CALL(hipEventRecord(end, stream));
    *flag = 1;
    CUDA_CALL(hipStreamSynchronize(stream));
    
    float ms;
    CUDA_CALL(hipEventElapsedTime(&ms, start, end));
    cout << ms * 1000 / r << "\n";

    CUDA_CALL(hipFree(arr));
    CUDA_CALL(hipFree(result));
    CUDA_CALL(hipHostFree((void*)flag));
    CUDA_CALL(hipStreamDestroy(stream));
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(end));
}